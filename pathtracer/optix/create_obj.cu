#include "hip/hip_runtime.h"
#include <include/optix/util.h>
#include "../material/conductor.h"
#include "../material/dielectric.h"
#include "../material/diffuse.h"
#include "../material/emitter.h"


/** 
 * \brief Initialize object on device.
 * 
 * \note Initailization must be excecuted only once.
 */
template <typename T, typename... Args>
__global__ void setup_object(T** d_ptr, Args... args) {
    (*d_ptr) = new T(args...);
}

template <typename T>
__global__ void delete_object(T* d_ptr) {
    delete d_ptr;
}

void pt::Conductor::setup_on_device() {
    hipMalloc(reinterpret_cast<void**>(&d_ptr), sizeof(Material*));
    setup_object<<<1,1>>>((Conductor**)&d_ptr, m_albedo, m_fuzz);
}

void pt::Conductor::delete_on_device() {
    delete_object<<<1,1>>>(d_ptr);
}

void pt::Dielectric::setup_on_device() {
    hipMalloc(reinterpret_cast<void**>(&d_ptr), sizeof(Material*));
    setup_object<<<1,1>>>((Dielectric**)&d_ptr, m_albedo, m_ior);
}

void pt::Dielectric::delete_on_device() {
    delete_object<<<1,1>>>(d_ptr);
}

void pt::Diffuse::setup_on_device() {
    hipMalloc(reinterpret_cast<void**>(&d_ptr), sizeof(Material*));
    setup_object<<<1,1>>>((Diffuse**)&d_ptr, m_albedo);
}

void pt::Diffuse::delete_on_device() {
    delete_object<<<1,1>>>(d_ptr);
}

void pt::Emitter::setup_on_device() {
    hipMalloc(reinterpret_cast<void**>(&d_ptr), sizeof(Material*));
    setup_object<<<1,1>>>((Emitter**)&d_ptr, m_color, m_strength);
}

void pt::Emitter::delete_on_device() {
    delete_object<<<1,1>>>(d_ptr);
}
