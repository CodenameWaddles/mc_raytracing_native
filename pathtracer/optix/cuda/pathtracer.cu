#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 

#include <optix.h>
#include <cuda/random.h>
#include "../../optix/util.h"
#include "../../optix/sbt.h"
#include "../../core/pathtracer.h"
#include "../../shape/optix/sphere.cuh"
#include "../../shape/optix/trianglemesh.cuh"
#include "../../material/conductor.h"
#include "../../material/dielectric.h"
#include "../../material/diffuse.h"
#include "../../material/emitter.h"

extern "C" {
__constant__ pt::Params params;
}

INLINE DEVICE bool trace_occlusion(
    OptixTraversableHandle handle, float3 ro, float3 rd, float tmin, float tmax
) 
{
    unsigned int occluded = 0u;
    optixTrace(
        handle, 
        ro, 
        rd, 
        tmin, 
        tmax,
        0.0f,
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
        RAY_TYPE_OCCLUSION,
        RAY_TYPE_COUNT,
        RAY_TYPE_OCCLUSION,
        occluded
    );
    return occluded;
}

INLINE DEVICE void trace_radiance(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax,
    pt::SurfaceInteraction*    si
) 
{
    // TODO: deduce stride from num ray-types passed in params

    unsigned int u0, u1;
    pack_pointer( si, u0, u1 );
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                // rayTime
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        RAY_TYPE_RADIANCE,        // SBT offset
        RAY_TYPE_COUNT,           // SBT stride
        RAY_TYPE_RADIANCE,        // missSBTIndex
        u0, u1 );	
}

// -------------------------------------------------------------------------------
static __forceinline__ __device__ void setPayloadOcclusion(bool occluded)
{
	optixSetPayload_0(static_cast<unsigned int>(occluded));
}

// -------------------------------------------------------------------------------
CALLABLE_FUNC void RG_FUNC(raygen)()
{
	const int w = params.width;
	const int h = params.height;
	const float3 eye = params.eye;
	const float3 U = params.U;
	const float3 V = params.V;
	const float3 W = params.W;
	const uint3 idx = optixGetLaunchIndex();
	const int subframe_index = params.subframe_index;

	unsigned int seed = tea<4>(idx.y * w + idx.x, subframe_index);

	float3 result = make_float3(0.0f, 0.0f, 0.0f);
	int i = params.samples_per_launch;

	do
	{
		const float2 subpixel_jitter = make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f);

		const float2 d = 2.0f * make_float2(
			(static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(w),
			(static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(h)
		) - 1.0f;
		float3 ray_direction = normalize(d.x * U + d.y * V + W);
		float3 ray_origin = eye;

		pt::SurfaceInteraction si;
		si.seed = seed;
		si.emission = make_float3(0.0f);
		si.radiance = make_float3(0.0f);
		si.attenuation = make_float3(1.0f);
		si.trace_terminate = false;

		float3 throughput = make_float3(1.0f);

		int depth = 0;
		for ( ;; ) {
			trace_radiance(
				params.handle,
				ray_origin, 
				ray_direction, 
				0.01f, 
				1e16f, 
				&si 
			);
	
			if ( si.trace_terminate || depth >= params.max_depth ) {
				result += si.emission * throughput;
				break;
			}
			
			throughput += si.emission;
			throughput *= si.attenuation;
			
			ray_origin = si.p;
			ray_direction = si.wo;

			++depth;
		}
	} while (--i);

	const uint3 launch_index = optixGetLaunchIndex();
	const unsigned int image_index = launch_index.y * params.width + launch_index.x;
	float3 accum_color = result / static_cast<float>(params.samples_per_launch);

	if (subframe_index > 0)
	{
		const float a = 1.0f / static_cast<float>(subframe_index + 1);
		const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
		accum_color = lerp(accum_color_prev, accum_color, a);
	}
	params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
	params.frame_buffer[image_index] = make_color(accum_color);
}

// -------------------------------------------------------------------------------
CALLABLE_FUNC void MS_FUNC(radiance)()
{
	pt::MissData* rt_data = reinterpret_cast<pt::MissData*>(optixGetSbtDataPointer());
	pt::SurfaceInteraction *si = get_surfaceinteraction();

	// si->radiance = make_float3(rt_data->bg_color);
	si->emission = make_float3(rt_data->bg_color);
	si->trace_terminate = true;
}

// -------------------------------------------------------------------------------
CALLABLE_FUNC void CH_FUNC(occlusion) ()
{
	setPayloadOcclusion(true);
}
