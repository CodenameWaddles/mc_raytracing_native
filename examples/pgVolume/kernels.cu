#include "hip/hip_runtime.h"
#include <prayground/prayground.h>
#include "params.h"

extern "C" { __constant__ LaunchParams params; }

using SurfaceInteraction = SurfaceInteraction_<float3>;

static __forceinline__ __device__ SurfaceInteraction* getSurfaceInteraction()
{
    const unsigned int u0 = getPayload<0>();
    const unsigned int u1 = getPayload<1>();
    return reinterpret_cast<SurfaceInteraction*>(unpackPointer(u0, u1));
}

static __forceinline__ __device__ void trace(
    OptixTraversableHandle handle,
    const float3& ro, const float3& rd,
    float tmin, float tmax,
    unsigned int ray_type,
    SurfaceInteraction* si
)
{
    unsigned int u0, u1;
    packPointer(si, u0, u1);
    optixTrace(
        handle,
        ro, rd,
        tmin, tmax, 0.0f,
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        ray_type,
        1,
        ray_type,
        u0, u1
    );
}

/* Raygen function */
static __forceinline__ __device__ void getCameraRay(
    const Camera::Data& camera,
    const float& x, const float& y,
    float3& ro, float3& rd)
{
    rd = normalize(x * camera.U + y * camera.V + camera.W);
    ro = camera.origin;
}

extern "C" __device__ void __raygen__medium()
{
    const RaygenData* raygen = reinterpret_cast<RaygenData*>(optixGetSbtDataPointer());

    const int subframe_index = params.frame;
    const uint3 idx = optixGetLaunchIndex();
    unsigned int seed = tea<4>(idx.x * params.width + idx.y, subframe_index);

    float3 result = make_float3(0.0f);
    float3 normal = make_float3(0.0f);
    float p_depth = 0.0f;
    float3 albedo = make_float3(0.0f);

    int i = params.samples_per_launch;

    do
    {
        const float2 subpixel_jitter = make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f);

        const float2 d = 2.0f * make_float2(
            (static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(params.width),
            (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(params.height)
        ) - 1.0f;

        float3 ro, rd;
        getCameraRay(raygen->camera, d.x, d.y, ro, rd);

        float3 throughput = make_float3(1.0f);

        SurfaceInteraction si;
        si.seed = seed;
        si.emission = make_float3(0.0f);
        si.albedo = make_float3(0.0f);
        si.trace_terminate = false;
        si.radiance_evaled = false;

        int depth = 0;
        for (;; ) {

            if (depth >= params.max_depth)
                break;

            trace(params.handle, ro, rd, 0.01f, 1e16f, 0, &si);

            if (si.trace_terminate) {
                result += si.emission * throughput;
                break;
            }

            // Get emission from area emitter
            if (si.surface_info.type == SurfaceType::AreaEmitter)
            {
                // Evaluating emission from emitter
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.bsdf_id,
                    &si,
                    si.surface_info.data
                    );
                result += si.emission * throughput;

                if (si.trace_terminate)
                    break;
            }
            // Specular sampling
            else if (+(si.surface_info.type & SurfaceType::Delta))
            {
                // Sampling scattered direction
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.sample_id,
                    &si,
                    si.surface_info.data
                    );

                // Evaluate bsdf
                float3 bsdf_val = optixContinuationCall<float3, SurfaceInteraction*, void*>(
                    si.surface_info.bsdf_id,
                    &si,
                    si.surface_info.data
                    );
                throughput *= bsdf_val;
            }
            // Rough surface sampling with applying MIS
            else if (+(si.surface_info.type & (SurfaceType::Rough | SurfaceType::Diffuse)))
            {
                // Importance sampling according to the BSDF
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.sample_id,
                    &si,
                    si.surface_info.data
                    );

                // Evaluate PDF depends on BSDF
                float bsdf_pdf = optixDirectCall<float, SurfaceInteraction*, void*>(
                    si.surface_info.pdf_id,
                    &si,
                    si.surface_info.data
                    );

                // Evaluate BSDF
                float3 bsdf_val = optixContinuationCall<float3, SurfaceInteraction*, void*>(
                    si.surface_info.bsdf_id,
                    &si,
                    si.surface_info.data
                    );

                throughput *= bsdf_val / bsdf_pdf;
            }

            ro = si.p;
            rd = si.wo;

            ++depth;
        }
    } while (--i);

    const uint3 launch_index = optixGetLaunchIndex();
    const unsigned int image_index = launch_index.y * params.width + launch_index.x;

    if (result.x != result.x) result.x = 0.0f;
    if (result.y != result.y) result.y = 0.0f;
    if (result.z != result.z) result.z = 0.0f;

    float3 accum_color = result / static_cast<float>(params.samples_per_launch);

    if (subframe_index > 0)
    {
        const float a = 1.0f / static_cast<float>(subframe_index + 1);
        const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
    uchar3 color = make_color(accum_color);
    params.result_buffer[image_index] = make_uchar4(color.x, color.y, color.z, 255);
}

/* Material functions */

/* Hitgroup functions */
extern "C" __device__ void __intersection__plane()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const Plane::Data* plane = reinterpret_cast<Plane::Data*>(data->shape_data);

    const float2 min = plane->min;
    const float2 max = plane->max;

    Ray ray = getLocalRay();

    const float t = -ray.o.y / ray.d.y;

    const float x = ray.o.x + t * ray.d.x;
    const float z = ray.o.z + t * ray.d.z;

    float2 uv = make_float2((x - min.x) / (max.x - min.x), (z - min.y) / (max.y - min.y));

    if (min.x < x && x < max.x && min.y < z && z < max.y && ray.tmin < t && t < ray.tmax)
        optixReportIntersection(t, 0, float2_as_ints(uv));
}

extern "C" __device__ void __closesthit__plane()
{
    HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());

    Ray ray = getWorldRay();

    float3 local_n = make_float3(0, 1, 0);
    const float3 world_n = normalize(optixTransformNormalFromObjectToWorldSpace(local_n));
    const float2 uv = getFloat2FromAttribute<0>();

    SurfaceInteraction* si = getSurfaceInteraction();

    si->p = ray.at(ray.tmax);
    si->shading.n = world_n;
    si->t = ray.tmax;
    si->wo = ray.d;
    si->uv = uv;
    si->surface_info = data->surface_info;
    si->shading.dpdu = optixTransformNormalFromObjectToWorldSpace(make_float3(1.0f, 0.0f, 0.0f));
    si->shading.dpdv = optixTransformNormalFromObjectToWorldSpace(make_float3(0.0f, 0.0f, 1.0f));
}

extern "C" __device__ void __intersection__grid()
{
    const HitgroupData* data = reinterpret_cast<const HitgroupData*>(optixGetSbtDataPointer());
    const GridMedium::Data* grid = reinterpret_cast<const GridMedium::Data*>(data->shape_data);
    const nanovdb::FloatGrid* density = reinterpret_cast<const nanovdb::FloatGrid*>(grid->density);
    assert(density);

    Ray ray = getLocalRay;

    auto bbox = density->indexBBox();
    float t0 = ray.tmin;
    float t1 = ray.tmax;
    auto iRay = nanovdb::Ray<float>(reinterpret_cast<const nanovdb::Vec3f&>(ray.o),
        reinterpret_cast<const nanovdb::Vec3f&>(ray.d), t0, t1);

    if (iRay.intersects(bbox, t0, t1))
    {
        optixSetPayload_2(__float_as_int(t1));
        optixReportIntersection(fmaxf(t0, ray.tmin), 0);
    }
}

extern "C" __device__ void __closesthit__grid()
{
    const HitgroupData* data = reinterpret_cast<const HitgroupData*>(optixGetSbtDataPointer());
    const GridMedium::Data* grid = reinterpret_cast<const GridMedium::Data*>(data->shape_data);
    const nanovdb::FloatGrid* density = reinterpret_cast<const nanovdb::FloatGrid*>(grid->density);
    const auto& tree = density->tree();
    auto acc = tree.getAccessor();

    Ray ray = getWorldRay();
    const float t0 = optixGetRayTmax();
    const float t1 = __int_as_float(optixGetPayload_2());
}

/* Texture functions */
extern "C" __device__ float3 __direct_callable__bitmap(const float2& uv, void* tex_data)
{
    const BitmapTexture::Data* image = reinterpret_cast<BitmapTexture::Data*>(tex_data);
    float4 c = tex2D<float4>(image->texture, uv.x, uv.y);
    return make_float3(c.x, c.y, c.z);
}

extern "C" __device__ float3 __direct_callable__constant(const float2& uv, void* tex_data)
{
    const ConstantTexture::Data* constant = reinterpret_cast<ConstantTexture::Data*>(tex_data);
    return constant->color;
}

extern "C" __device__ float3 __direct_callable__checker(const float2& uv, void* tex_data)
{
    const CheckerTexture::Data* checker = reinterpret_cast<CheckerTexture::Data*>(tex_data);
    const bool is_odd = sinf(uv.x * math::pi * checker->scale) * sinf(uv.y * math::pi * checker->scale);
    return is_odd ? checker->color1 : checker->color2;
}



