#include "hip/hip_runtime.h"
#include "util.cuh"
#include <prayground/shape/plane.h>
#include <prayground/shape/trianglemesh.h>
#include <prayground/shape/sphere.h>
#include <prayground/shape/cylinder.h>
#include <prayground/core/ray.h>
#include <prayground/core/onb.h>
#include <prayground/core/bsdf.h>

using namespace prayground;

extern "C" __device__ void __closesthit__shadow()
{
    optixSetPayload_0(1);
}

// Plane -------------------------------------------------------------------------------
static __forceinline__ __device__ bool hitPlane(const PlaneData* plane_data, const float3& o, const float3& v, const float tmin, const float tmax, SurfaceInteraction& si)
{
    const float2 min = plane_data->min;
    const float2 max = plane_data->max;
    
    const float t = -o.y / v.y;
    const float x = o.x + t * v.x;
    const float z = o.z + t * v.z;

    if (min.x < x && x < max.x && min.y < z && z < max.y && tmin < t && t < tmax)
    {
        si.uv = make_float2((x - min.x) / (max.x - min.x), (z - min.y) / max.y - min.y);
        si.n = make_float3(0, 1, 0);
        si.t = t;
        si.p = o + t*v;
        return true;
    }
    return false;
}

extern "C" __device__ void __intersection__plane()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const PlaneData* plane_data = reinterpret_cast<PlaneData*>(data->shape_data);

    const float2 min = plane_data->min;
    const float2 max = plane_data->max;

    Ray ray = getLocalRay();

    const float t = -ray.o.y / ray.d.y;

    const float x = ray.o.x + t * ray.d.x;
    const float z = ray.o.z + t * ray.d.z;

    float2 uv = make_float2((x - min.x) / (max.x - min.x), (z - min.y) / (max.y - min.y));

    if (min.x < x && x < max.x && min.y < z && z < max.y && ray.tmin < t && t < ray.tmax)
        optixReportIntersection(t, 0, float2_as_ints(uv));
}

extern "C" __device__ void __closesthit__plane()
{
    HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());

    Ray ray = getWorldRay();

    float3 local_n = make_float3(0, 1, 0);
    float3 world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);
    float2 uv = make_float2(
        int_as_float( optixGetAttribute_0() ), 
        int_as_float( optixGetAttribute_1() )
    );

    SurfaceInteraction* si = getSurfaceInteraction();

    si->p = ray.at(ray.tmax);
    si->n = world_n;
    si->t = ray.tmax;
    si->wi = ray.d;
    si->uv = uv;
    si->surface_info = data->surface_info;
}

extern "C" __device__ float __continuation_callable__pdf_plane(AreaEmitterInfo area_info, const float3 & origin, const float3 & direction)
{
    const PlaneData* plane_data = reinterpret_cast<PlaneData*>(area_info.shape_data);

    SurfaceInteraction si;
    const float3 local_o = area_info.worldToObj.pointMul(origin);
    const float3 local_d = area_info.worldToObj.vectorMul(direction);

    if (!hitPlane(plane_data, local_o, local_d, 0.01f, 1e16f, si))
        return 0.0f;

    const float3 corner0 = area_info.objToWorld.pointMul(make_float3(plane_data->min.x, 0.0f, plane_data->min.y));
    const float3 corner1 = area_info.objToWorld.pointMul(make_float3(plane_data->max.x, 0.0f, plane_data->min.y));
    const float3 corner2 = area_info.objToWorld.pointMul(make_float3(plane_data->min.x, 0.0f, plane_data->max.y));
    si.n = normalize(area_info.objToWorld.vectorMul(si.n));
    const float area = length(cross(corner1 - corner0, corner2 - corner0));
    const float distance_squared = si.t * si.t;
    const float cosine = fabs(dot(si.n, direction));
    if (cosine < math::eps)
        return 0.0f;
    return distance_squared / (cosine * area);
}

// Return light vector in global space from si.p to random light point
extern "C" __device__ float3 __direct_callable__rnd_sample_plane(AreaEmitterInfo area_info, SurfaceInteraction * si)
{
    const PlaneData* plane_data = reinterpret_cast<PlaneData*>(area_info.shape_data);
    // Transform point from world to object space
    const float3 local_p = area_info.worldToObj.pointMul(si->p);
    unsigned int seed = si->seed;
    // Get random point on area emitter
    const float3 rnd_p = make_float3(rnd(seed, plane_data->min.x, plane_data->max.x), 0.0f, rnd(seed, plane_data->min.y, plane_data->max.y));
    float3 to_light = rnd_p - local_p;
    to_light = area_info.objToWorld.vectorMul(to_light);
    si->seed = seed;
    return to_light;
}

// Sphere -------------------------------------------------------------------------------
static __forceinline__ __device__ float2 getSphereUV(const float3& p) {
    float phi = atan2(p.z, p.x);
    float theta = asin(p.y);
    float u = 1.0f - (phi + math::pi) / (2.0f * math::pi);
    float v = 1.0f - (theta + math::pi / 2.0f) / math::pi;
    return make_float2(u, v);
}

static __forceinline__ __device__ bool hitSphere(const SphereData* sphere_data, const float3& o, const float3& v, const float tmin, const float tmax, SurfaceInteraction& si)
{
    const float3 center = sphere_data->center;
    const float radius = sphere_data->radius;

    const float3 oc = o - center;
    const float a = dot(v, v);
    const float half_b = dot(oc, v);
    const float c = dot(oc, oc) - radius * radius;
    const float discriminant = half_b * half_b - a * c;

    if (discriminant <= 0.0f) return false;

    const float sqrtd = sqrtf(discriminant);

    float t = (-half_b - sqrtd) / a;
    if (t < tmin || tmax < t)
    {
        t = (-half_b + sqrtd) / a;
        if (t < tmin || tmax < t)
            return false;
    }

    si.t = t;
    si.p = o + t * v;
    si.n = si.p / radius;
    si.uv = getSphereUV(si.n);
    return true;
}

extern "C" __device__ void __intersection__sphere() {
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const SphereData* sphere_data = reinterpret_cast<SphereData*>(data->shape_data);

    const float3 center = sphere_data->center;
    const float radius = sphere_data->radius;

    Ray ray = getLocalRay();

    const float3 oc = ray.o - center;
    const float a = dot(ray.d, ray.d);
    const float half_b = dot(oc, ray.d);
    const float c = dot(oc, oc) - radius * radius;
    const float discriminant = half_b * half_b - a * c;

    if (discriminant > 0.0f) {
        float sqrtd = sqrtf(discriminant);
        float t1 = (-half_b - sqrtd) / a;
        bool check_second = true;
        if (t1 > ray.tmin && t1 < ray.tmax) {
            float3 normal = normalize((ray.at(t1) - center) / radius);
            check_second = false;
            optixReportIntersection(t1, 0, float3_as_ints(normal));
        }

        if (check_second) {
            float t2 = (-half_b + sqrtd) / a;
            if (t2 > ray.tmin && t2 < ray.tmax) {
                float3 normal = normalize((ray.at(t2) - center) / radius);
                optixReportIntersection(t2, 0, float3_as_ints(normal));
            }
        }
    }
}

extern "C" __device__ void __closesthit__sphere() {
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const SphereData* sphere_data = reinterpret_cast<SphereData*>(data->shape_data);

    Ray ray = getWorldRay();

    float3 local_n = make_float3(
        int_as_float(optixGetAttribute_0()),
        int_as_float(optixGetAttribute_1()),
        int_as_float(optixGetAttribute_2())
    );
    float3 world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);

    SurfaceInteraction* si = getSurfaceInteraction();
    si->p = ray.at(ray.tmax);
    si->n = world_n;
    si->t = ray.tmax;
    si->wi = ray.d;
    si->uv = getSphereUV(local_n);
    si->surface_info = data->surface_info;
}

// Cylinder -------------------------------------------------------------------------------
static INLINE DEVICE float2 getCylinderUV(
    const float3& p, const float radius, const float height, const bool hit_disk
)
{
    if (hit_disk)
    {
        const float r = sqrtf(p.x*p.x + p.z*p.z) / radius;
        const float theta = atan2(p.z, p.x);
        float u = 1.0f - (theta + math::pi/2.0f) / math::pi;
        return make_float2(u, r);
    } 
    else
    {
        const float theta = atan2(p.z, p.x);
        const float v = (p.y + height / 2.0f) / height;
        float u = 1.0f - (theta + math::pi/2.0f) / math::pi;
        return make_float2(u, v);
    }
}

extern "C" __device__ void __intersection__cylinder()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const CylinderData* cylinder = reinterpret_cast<CylinderData*>(data->shape_data);

    const float radius = cylinder->radius;
    const float height = cylinder->height;

    Ray ray = getLocalRay();
    
    const float a = dot(ray.d, ray.d) - ray.d.y * ray.d.y;
    const float half_b = (ray.o.x * ray.d.x + ray.o.z * ray.d.z);
    const float c = dot(ray.o, ray.o) - ray.o.y * ray.o.y - radius*radius;
    const float discriminant = half_b*half_b - a*c;

    if (discriminant > 0.0f)
    {
        const float sqrtd = sqrtf(discriminant);
        const float side_t1 = (-half_b - sqrtd) / a;
        const float side_t2 = (-half_b + sqrtd) / a;

        const float side_tmin = fmin( side_t1, side_t2 );
        const float side_tmax = fmax( side_t1, side_t2 );

        if ( side_tmin > ray.tmax || side_tmax < ray.tmin )
            return;

        const float upper = height / 2.0f;
        const float lower = -height / 2.0f;
        const float y_tmin = fmin( (lower - ray.o.y) / ray.d.y, (upper - ray.o.y) / ray.d.y );
        const float y_tmax = fmax( (lower - ray.o.y) / ray.d.y, (upper - ray.o.y) / ray.d.y );

        float t1 = fmax(y_tmin, side_tmin);
        float t2 = fmin(y_tmax, side_tmax);
        if (t1 > t2 || (t2 < ray.tmin) || (t1 > ray.tmax))
            return;
        
        bool check_second = true;
        if (ray.tmin < t1 && t1 < ray.tmax)
        {
            float3 P = ray.at(t1);
            bool hit_disk = y_tmin > side_tmin;
            float3 normal = hit_disk 
                          ? normalize(P - make_float3(P.x, 0.0f, P.z))   // Hit at disk
                          : normalize(P - make_float3(0.0f, P.y, 0.0f)); // Hit at side
            float2 uv = getCylinderUV(P, radius, height, hit_disk);
            optixReportIntersection(t1, 0, float3_as_ints(normal), float2_as_ints(uv));
            check_second = false;
        }
        
        if (check_second)
        {
            if (ray.tmin < t2 && t2 < ray.tmax)
            {
                float3 P = ray.at(t2);
                bool hit_disk = y_tmax < side_tmax;
                float3 normal = hit_disk
                            ? normalize(P - make_float3(P.x, 0.0f, P.z))   // Hit at disk
                            : normalize(P - make_float3(0.0f, P.y, 0.0f)); // Hit at side
                float2 uv = getCylinderUV(P, radius, height, hit_disk);
                optixReportIntersection(t2, 0, float3_as_ints(normal), float2_as_ints(uv));
            }
        }
    }
}

extern "C" __device__ void __closesthit__cylinder()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const CylinderData* cylinder = reinterpret_cast<CylinderData*>(data->shape_data);

    Ray ray = getWorldRay();

    float3 local_n = make_float3(
        int_as_float( optixGetAttribute_0() ),
        int_as_float( optixGetAttribute_1() ), 
        int_as_float( optixGetAttribute_2() )
    );

    float2 uv = make_float2(
        int_as_float( optixGetAttribute_3() ),
        int_as_float( optixGetAttribute_4() )
    );

    float3 world_n = optixTransformNormalFromObjectToWorldSpace(local_n);

    SurfaceInteraction* si = getSurfaceInteraction();
    si->p = ray.at(ray.tmax);
    si->n = normalize(world_n);
    si->t = ray.tmax;
    si->wi = ray.d;
    si->uv = uv;
    si->surface_info = data->surface_info;
}

// Triangle mesh -------------------------------------------------------------------------------
extern "C" __device__ void __closesthit__mesh()
{
    HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const MeshData* mesh_data = reinterpret_cast<MeshData*>(data->shape_data);

    Ray ray = getWorldRay();
    
    const int prim_id = optixGetPrimitiveIndex();
    const Face face = mesh_data->faces[prim_id];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    const float2 texcoord0 = mesh_data->texcoords[face.texcoord_id.x];
    const float2 texcoord1 = mesh_data->texcoords[face.texcoord_id.y];
    const float2 texcoord2 = mesh_data->texcoords[face.texcoord_id.z];
    const float2 texcoords = (1-u-v)*texcoord0 + u*texcoord1 + v*texcoord2;

    float3 n0 = mesh_data->normals[face.normal_id.x];
	float3 n1 = mesh_data->normals[face.normal_id.y];
	float3 n2 = mesh_data->normals[face.normal_id.z];

    // Linear interpolation of normal by barycentric coordinates.
    float3 local_n = (1.0f-u-v)*n0 + u*n1 + v*n2;
    float3 world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);

    SurfaceInteraction* si = getSurfaceInteraction();
    si->p = ray.at(ray.tmax);
    si->n = world_n;
    si->t = ray.tmax;
    si->wi = ray.d;
    si->uv = texcoords;
    si->surface_info = data->surface_info;
}