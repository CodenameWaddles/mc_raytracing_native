#include "hip/hip_runtime.h"
#include "util.cuh"
#include <prayground/texture/bitmap.h>
#include <prayground/texture/constant.h>
#include <prayground/texture/checker.h>

using namespace prayground;

extern "C" __device__ float4 __direct_callable__bitmap(const float2& uv, void* tex_data) {
    const BitmapTextureData* image = reinterpret_cast<BitmapTextureData*>(tex_data);
    float4 c = tex2D<float4>(image->texture, uv.x, uv.y);
    return c;
}

extern "C" __device__ float4 __direct_callable__constant(const float2& uv, void* tex_data) {
    const ConstantTextureData* constant = reinterpret_cast<ConstantTextureData*>(tex_data);
    return constant->color;
}

extern "C" __device__ float4 __direct_callable__checker(const float2& uv, void* tex_data) {
    const CheckerTextureData* checker = reinterpret_cast<CheckerTextureData*>(tex_data);
    const bool is_odd = sinf(uv.x*M_PIf*checker->scale) * sinf(uv.y*M_PIf*checker->scale) < 0;
    return is_odd ? checker->color1 : checker->color2;
}