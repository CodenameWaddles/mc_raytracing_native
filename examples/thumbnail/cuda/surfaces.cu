#include "hip/hip_runtime.h"
#include "util.cuh"

// Diffuse -----------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_diffuse(SurfaceInteraction* si, void* mat_data) {
    const Diffuse::Data* diffuse = reinterpret_cast<Diffuse::Data*>(mat_data);

    if (diffuse->twosided)
        si->shading.n = faceforward(si->shading.n, -si->wo, si->shading.n);

    si->trace_terminate = false;
    uint32_t seed = si->seed;
    const Vec2f u = UniformSampler::get2D(seed);
    Vec3f wi = cosineSampleHemisphere(u[0], u[1]);
    Onb onb(si->shading.n);
    onb.inverseTransform(wi);
    si->wi = normalize(wi);
    si->seed = seed;
}

extern "C" __device__ Vec3f __continuation_callable__bsdf_diffuse(SurfaceInteraction* si, void* mat_data)
{
    const Diffuse::Data* diffuse = reinterpret_cast<Diffuse::Data*>(mat_data);
    const Vec4f albedo = optixDirectCall<Vec4f, const Vec2f&, void*>(diffuse->texture.prg_id, si->shading.uv, diffuse->texture.data);
    si->albedo = Vec3f(albedo);
    si->emission = Vec3f(0.0f);
    const float cosine = fmaxf(0.0f, dot(si->shading.n, si->wi));
    return si->albedo * cosine * math::inv_pi;
}

extern "C" __device__ float __direct_callable__pdf_diffuse(SurfaceInteraction* si, void* mat_data)
{
    const float cosine = fmaxf(0.0f, dot(si->shading.n, si->wi));
    return cosine * math::inv_pi;
}

// Dielectric --------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_dielectric(SurfaceInteraction* si, void* mat_data) {
    const Dielectric::Data* dielectric = reinterpret_cast<Dielectric::Data*>(mat_data);

    float ni = 1.000292f; // air
    float nt = dielectric->ior;  // ior specified 
    float cosine = dot(si->wo, si->shading.n);
    bool into = cosine < 0;
    Vec3f outward_normal = into ? si->shading.n : -si->shading.n;

    if (!into) swap(ni, nt);

    cosine = fabs(cosine);
    float sine = sqrtf(1.0 - cosine*cosine);
    bool cannot_refract = (ni / nt) * sine > 1.0f;

    float reflect_prob = fresnel(cosine, ni, nt);
    unsigned int seed = si->seed;

    if (cannot_refract || reflect_prob > rnd(seed))
        si->wi = reflect(si->wo, outward_normal);
    else    
        si->wi = refract(si->wo, outward_normal, cosine, ni, nt);
    si->radiance_evaled = false;
    si->trace_terminate = false;
    si->seed = seed;
}

extern "C" __device__ Vec3f __continuation_callable__bsdf_dielectric(SurfaceInteraction* si, void* mat_data)
{
    const Dielectric::Data* dielectric = reinterpret_cast<Dielectric::Data*>(mat_data);
    si->emission = Vec3f(0.0f);
    Vec4f albedo = optixDirectCall<Vec4f, const Vec2f&, void*>(dielectric->texture.prg_id, si->shading.uv, dielectric->texture.data);
    si->albedo = Vec3f(albedo);
    return si->albedo;
}

extern "C" __device__ float __direct_callable__pdf_dielectric(SurfaceInteraction* si, void* mat_data)
{
    return 1.0f;
}

// Conductor --------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_conductor(SurfaceInteraction* si, void* mat_data) {
    const Conductor::Data* conductor = reinterpret_cast<Conductor::Data*>(mat_data);
    if (conductor->twosided)
        si->shading.n = faceforward(si->shading.n, -si->wo, si->shading.n);

    si->wi = reflect(si->wo, si->shading.n);
    si->trace_terminate = false;
    si->radiance_evaled = false;
}

extern "C" __device__ Vec3f __continuation_callable__bsdf_conductor(SurfaceInteraction* si, void* mat_data)
{
    const Conductor::Data* conductor = reinterpret_cast<Conductor::Data*>(mat_data);
    si->emission = Vec3f(0.0f);
    Vec4f albedo = optixDirectCall<Vec4f, const Vec2f&, void*>(conductor->texture.prg_id, si->shading.uv, conductor->texture.data);
    si->albedo = Vec3f(albedo);
    return si->albedo;
}

extern "C" __device__ float __direct_callable__pdf_conductor(SurfaceInteraction* si, void* mat_data)
{
    return 1.0f;
}

// Disney BRDF ------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_disney(SurfaceInteraction* si, void* mat_data)
{
    const Disney::Data* disney = reinterpret_cast<Disney::Data*>(mat_data);

    if (disney->twosided)
        si->shading.n = faceforward(si->shading.n, -si->wo, si->shading.n);

    unsigned int seed = si->seed;
    const Vec2f u = UniformSampler::get2D(seed);
    const float diffuse_ratio = 0.5f * (1.0f - disney->metallic);
    Onb onb(si->shading.n);

    if (rnd(seed) < diffuse_ratio)
    {
        Vec3f wi = cosineSampleHemisphere(u[0], u[1]);
        onb.inverseTransform(wi);
        si->wi = normalize(wi);
    }
    else
    {
        float gtr2_ratio = 1.0f / (1.0f + disney->clearcoat);
        Vec3f h;
        const float alpha = fmaxf(0.001f, disney->roughness);
        /// For debugging microfacet sampling according to this cite @ref: https://jcgt.org/published/0007/04/01/
        //const float aspect = sqrtf(1.0f - disney->anisotropic * 0.9f);
        //const float ax = fmaxf(0.001f, math::sqr(alpha) / aspect);
        //const float ay = fmaxf(0.001f, math::sqr(alpha) * aspect);
        if (rnd(seed) < gtr2_ratio)
             h = sampleGGX(u[0], u[1], alpha);
            //h = sampleGGXAniso(-si->wo, ax, ay, z1, z2);
        else
            h = sampleGTR1(u[0], u[1], alpha);
        onb.inverseTransform(h);
        si->wi = normalize(reflect(si->wo, h));
    }
    si->radiance_evaled = false;
    si->trace_terminate = false;
    si->seed = seed;
}

/**
 * @ref: https://rayspace.xyz/CG/contents/Disney_principled_BRDF/
 * 
 * @note 
 * ===== Prefix =====
 * F : fresnel 
 * f : brdf function
 * G : geometry function
 * D : normal distribution function
 */
extern "C" __device__ Vec3f __continuation_callable__bsdf_disney(SurfaceInteraction* si, void* mat_data)
{   
    const Disney::Data* disney = reinterpret_cast<Disney::Data*>(mat_data);
    si->emission = Vec3f(0.0f);

    const Vec3f V = -normalize(si->wo);
    const Vec3f L = normalize(si->wi);
    const Vec3f N = normalize(si->shading.n);

    const float NdotV = dot(N, V);
    const float NdotL = dot(N, L);

    if (NdotV <= 0.0f || NdotL <= 0.0f)
        return Vec3f(0.0f);

    const Vec3f H = normalize(V + L);
    const float NdotH = dot(N, H);
    const float LdotH /* = VdotH */ = dot(L, H);

    Vec4f tmp = optixDirectCall<Vec4f, const Vec2f&, void*>(
        disney->base.prg_id, si->shading.uv, disney->base.data);
    const Vec3f base_color = Vec3f(tmp);
    si->albedo = base_color;

    // Diffuse term (diffuse, subsurface, sheen) ======================
    // Diffuse
    const float Fd90 = 0.5f + 2.0f * disney->roughness * LdotH*LdotH;
    const float FVd90 = fresnelSchlickT(NdotV, Fd90);
    const float FLd90 = fresnelSchlickT(NdotL, Fd90);
    const Vec3f f_diffuse = (base_color / math::pi) * FVd90 * FLd90;

    // Subsurface
    const float Fss90 = disney->roughness * LdotH*LdotH;
    const float FVss90 = fresnelSchlickT(NdotV, Fss90);
    const float FLss90 = fresnelSchlickT(NdotL, Fss90); 
    const Vec3f f_subsurface = (base_color / math::pi) * 1.25f * (FVss90 * FLss90 * ((1.0f / (NdotV * NdotL)) - 0.5f) + 0.5f);

    // Sheen
    const Vec3f rho_tint = base_color / luminance(base_color);
    const Vec3f rho_sheen = lerp(Vec3f(1.0f), rho_tint, disney->sheen_tint);
    const Vec3f f_sheen = disney->sheen * rho_sheen * powf(1.0f - LdotH, 5.0f);

    // Specular term (specular, clearcoat) ============================
    // Spcular
    const Vec3f X = si->shading.dpdu;
    const Vec3f Y = si->shading.dpdv;
    const float alpha = fmaxf(0.001f, disney->roughness);
    const float aspect = sqrtf(1.0f - disney->anisotropic * 0.9f);
    const float ax = fmaxf(0.001f, pow2(alpha) / aspect);
    const float ay = fmaxf(0.001f, pow2(alpha) * aspect);
    const Vec3f rho_specular = lerp(Vec3f(1.0f), rho_tint, disney->specular_tint);
    const Vec3f Fs0 = lerp(0.08f * disney->specular * rho_specular, base_color, disney->metallic);
    const Vec3f FHs0 = fresnelSchlickR(LdotH, Fs0);
    const float Ds = GTR2_aniso(NdotH, dot(H, X), dot(H, Y), ax, ay);
    float Gs = smithG_GGX_aniso(NdotL, dot(L, X), dot(L, Y), ax, ay);
    Gs *= smithG_GGX_aniso(NdotV, dot(V, X), dot(V, Y), ax, ay);
    const Vec3f f_specular = FHs0 * Ds * Gs;

    // Clearcoat
    const float Fcc = fresnelSchlickR(LdotH, 0.04f);
    const float alpha_cc = lerp(0.1f, 0.001f, disney->clearcoat_gloss);
    const float Dcc = GTR1(NdotH, alpha_cc);
    const float Gcc = smithG_GGX(NdotV, 0.25f);
    const Vec3f f_clearcoat = Vec3f( 0.25f * disney->clearcoat * (Fcc * Dcc * Gcc) );

    const Vec3f out = ( 1.0f - disney->metallic ) * ( lerp( f_diffuse, f_subsurface, disney->subsurface ) + f_sheen ) + f_specular + f_clearcoat;
    return out * clamp(NdotL, 0.0f, 1.0f);
}

/**
 * @ref http://simon-kallweit.me/rendercompo2015/report/#adaptivesampling
 * 
 * @todo Investigate correct evaluation of PDF.
 */
extern "C" __device__ float __direct_callable__pdf_disney(SurfaceInteraction* si, void* mat_data)
{
    const Disney::Data* disney = reinterpret_cast<Disney::Data*>(mat_data);

    const Vec3f V = -si->wo;
    const Vec3f L = si->wi;
    const Vec3f N = si->shading.n;

    const float diffuse_ratio = 0.5f * (1.0f - disney->metallic);
    const float specular_ratio = 1.0f - diffuse_ratio;

    const float NdotL = dot(N, L);
    const float NdotV = dot(N, V);

    if (NdotL <= 0.0f || NdotV <= 0.0f)
        return 1.0f;

    const float alpha = fmaxf(0.001f, disney->roughness);
    const float alpha_cc = lerp(0.1f, 0.001f, disney->clearcoat_gloss);
    const Vec3f H = normalize(V + L);
    const float NdotH = abs(dot(H, N));

    const float pdf_Ds = GTR2(NdotH, alpha);
    const float pdf_Dcc = GTR1(NdotH, alpha_cc);
    const float ratio = 1.0f / (1.0f + disney->clearcoat);
    const float pdf_specular = (pdf_Dcc + ratio * (pdf_Ds - pdf_Dcc));
    const float pdf_diffuse = NdotL * math::inv_pi;

    return diffuse_ratio * pdf_diffuse + specular_ratio * pdf_specular;
}

// Area emitter ------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__area_emitter(SurfaceInteraction* si, void* surface_data)
{
    const AreaEmitter::Data* area = reinterpret_cast<AreaEmitter::Data*>(surface_data);
    si->trace_terminate = true;
    float is_emitted = dot(si->wo, si->shading.n) < 0.0f ? 1.0f : 0.0f;
    if (area->twosided)
    {
        is_emitted = 1.0f;
        si->shading.n = faceforward(si->shading.n, -si->wo, si->shading.n);
    }

    const Vec4f base = optixDirectCall<Vec4f, const Vec2f&, void*>(
        area->texture.prg_id, si->shading.uv, area->texture.data);
    si->albedo = Vec3f(base);
    
    si->emission = si->albedo * area->intensity * is_emitted;
}
