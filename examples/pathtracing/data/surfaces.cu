#include "hip/hip_runtime.h"
#include "util.cuh"
#include <prayground/material/diffuse.h>
#include <prayground/material/dielectric.h>
#include <prayground/material/conductor.h>
#include <prayground/material/disney.h>
#include <prayground/emitter/area.h>
#include <prayground/core/bsdf.h>
#include <prayground/core/onb.h>
#include <prayground/core/color.h>

using namespace prayground;

/// @todo 確率密度関数、Next event estimationの実装

// Diffuse -----------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_diffuse(SurfaceInteraction* si, void* mat_data) {
    const DiffuseData* diffuse = reinterpret_cast<DiffuseData*>(mat_data);

    if (diffuse->twosided)
        si->n = faceforward(si->n, -si->wi, si->n);

    si->trace_terminate = false;
    unsigned int seed = si->seed;
    float3 wi = randomSampleHemisphere(seed);
    Onb onb(si->n);
    onb.inverseTransform(wi);
    si->wo = normalize(wi);
    si->seed = seed;
}

extern "C" __device__ float3 __continuation_callable__bsdf_diffuse(SurfaceInteraction* si, void* mat_data)
{
    const DiffuseData* diffuse = reinterpret_cast<DiffuseData*>(mat_data);
    const float3 albedo = optixDirectCall<float3, SurfaceInteraction*, void*>(diffuse->tex_program_id, si, diffuse->tex_data);
    si->albedo = albedo;
    const float cosine = fmaxf(0.0f, dot(si->n, si->wo));
    return albedo * (cosine / math::pi);
}

extern "C" __device__ float __direct_callable__pdf_diffuse(SurfaceInteraction* si, void* mat_data)
{
    const float cosine = fmaxf(0.0f, dot(si->n, si->wo));
    return cosine / math::pi;
}

// Dielectric --------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_dielectric(SurfaceInteraction* si, void* mat_data) {
    const DielectricData* dielectric = reinterpret_cast<DielectricData*>(mat_data);

    float ni = 1.0f; // air
    float nt = dielectric->ior;  // ior specified 
    float cosine = dot(si->wi, si->n);
    bool into = cosine < 0;
    float3 outward_normal = into ? si->n : -si->n;

    if (!into) swap(ni, nt);

    cosine = fabs(cosine);
    float sine = sqrtf(1.0 - cosine*cosine);
    bool cannot_refract = (ni / nt) * sine > 1.0f;

    float reflect_prob = fresnel(cosine, ni, nt);
    unsigned int seed = si->seed;

    if (cannot_refract || reflect_prob > rnd(seed))
        si->wo = reflect(si->wi, outward_normal);
    else    
        si->wo = refract(si->wi, outward_normal, cosine, ni, nt);
    si->radiance_evaled = false;
    si->trace_terminate = false;
    si->seed = seed;
}

extern "C" __device__ float3 __continuation_callable__bsdf_dielectric(SurfaceInteraction* si, void* mat_data)
{
    const DielectricData* dielectric = reinterpret_cast<DielectricData*>(mat_data);
    si->emission = make_float3(0.0f);
    float3 albedo = optixDirectCall<float3, SurfaceInteraction*, void*>(dielectric->tex_program_id, si, dielectric->tex_data);
    si->albedo = albedo;
    return albedo;
}

extern "C" __device__ float __direct_callable__pdf_dielectric(SurfaceInteraction* si, void* mat_data)
{
    return 1.0f;
}

// Conductor --------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_conductor(SurfaceInteraction* si, void* mat_data) {
    const ConductorData* conductor = reinterpret_cast<ConductorData*>(mat_data);
    if (conductor->twosided)
        si->n = faceforward(si->n, -si->wi, si->n);

    si->wo = reflect(si->wi, si->n);
    si->trace_terminate = false;
    si->radiance_evaled = false;
}

extern "C" __device__ float3 __continuation_callable__bsdf_conductor(SurfaceInteraction* si, void* mat_data)
{
    const ConductorData* conductor = reinterpret_cast<ConductorData*>(mat_data);
    si->emission = make_float3(0.0f);
    float3 albedo = optixDirectCall<float3, SurfaceInteraction*, void*>(conductor->tex_program_id, si, conductor->tex_data);
    si->albedo = albedo;
    return albedo;
}

extern "C" __device__ float __direct_callable__pdf_conductor(SurfaceInteraction* si, void* mat_data)
{
    return 1.0f;
}
// Disney BRDF ------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_disney(SurfaceInteraction* si, void* mat_data)
{
    const DisneyData* disney = reinterpret_cast<DisneyData*>(mat_data);

    if (disney->twosided)
        si->n = faceforward(si->n, -si->wi, si->n);

    unsigned int seed = si->seed;
    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    const float diffuse_ratio = 0.5f * (1.0f - disney->metallic);
    Onb onb(si->n);

    if (rnd(seed) < diffuse_ratio)
    {
        float3 w_in = cosineSampleHemisphere(z1, z2);
        onb.inverseTransform(w_in);
        si->wo = normalize(w_in);
    }
    else
    {
        /// @todo Change sampling functions according to the ratio to choose specular or clearcoat pdf
        /// ratio = 1.0f / (1.0f + clearcoat);

        float3 h = sampleGGX(z1, z2, disney->roughness);
        onb.inverseTransform(h);
        si->wo = normalize(reflect(si->wi, h));
    }
    si->radiance_evaled = false;
    si->trace_terminate = false;
    si->seed = seed;
}

/**
 * @ref: https://rayspace.xyz/CG/contents/Disney_principled_BRDF/
 * 
 * @note 
 * ===== Prefix =====
 * F : fresnel 
 * f : brdf function
 * G : geometry function
 * D : normal distribution function
 */
extern "C" __device__ float3 __continuation_callable__bsdf_disney(SurfaceInteraction* si, void* mat_data)
{   
    const DisneyData* disney = reinterpret_cast<DisneyData*>(mat_data);
    si->emission = make_float3(0.0f);

    const float3 V = -normalize(si->wi);
    const float3 L = normalize(si->wo);
    const float3 N = normalize(si->n);

    const float NdotV = fabs(dot(N, V));
    const float NdotL = fabs(dot(N, L));

    if (NdotV == 0.0f || NdotL == 0.0f)
        return make_float3(0.0f);

    const float3 H = normalize(V + L);
    const float NdotH = dot(N, H);
    const float LdotH /* = VdotH */ = dot(L, H);

    const float3 base_color = optixDirectCall<float3, SurfaceInteraction*, void*>(
        disney->base_program_id, si, disney->base_tex_data
    );
    si->albedo = base_color;

    // Diffuse term (diffuse, subsurface, sheen) ======================
    // Diffuse
    const float Fd90 = 0.5f + 2.0f * disney->roughness * LdotH*LdotH;
    const float FVd90 = fresnelSchlickT(NdotV, Fd90);
    const float FLd90 = fresnelSchlickT(NdotL, Fd90);
    const float3 f_diffuse = (base_color / math::pi) * FVd90 * FLd90;

    // Subsurface
    const float Fss90 = disney->roughness * LdotH*LdotH;
    const float FVss90 = fresnelSchlickT(NdotV, Fss90);
    const float FLss90 = fresnelSchlickT(NdotL, Fss90); 
    const float3 f_subsurface = (base_color / math::pi) * 1.25f * (FVss90 * FLss90 * ((1.0f / (NdotV * NdotL)) - 0.5f) + 0.5f);

    // Sheen
    const float3 rho_tint = base_color / luminance(base_color);
    const float3 rho_sheen = lerp(make_float3(1.0f), rho_tint, disney->sheen_tint);
    const float3 f_sheen = disney->sheen * rho_sheen * powf(1.0f - LdotH, 5.0f);

    // Specular term (specular, clearcoat) ============================
    // Spcular
    Onb onb(N);
    const float3 X = onb.tangent;
    const float3 Y = onb.bitangent;
    const float alpha = fmaxf(0.001f, disney->roughness * disney->roughness); // Remapping of roughness
    const float aspect = sqrtf(1.0f - disney->anisotropic * 0.9f);
    const float ax = fmaxf(0.001f, math::sqr(disney->roughness) / aspect);
    const float ay = fmaxf(0.001f, math::sqr(disney->roughness) * aspect);
    const float3 rho_specular = lerp(make_float3(1.0f), rho_tint, disney->specular_tint);
    const float3 Fs0 = lerp(0.08f * disney->specular * rho_specular, base_color, disney->metallic);
    const float3 FHs0 = fresnelSchlickR(LdotH, Fs0);
    const float Ds = GTR2_aniso(NdotH, dot(H, X), dot(H, Y), ax, ay);
    const float alpha_g = powf(0.5f*disney->roughness + 0.5f, 2.0f);
    const float Gs = geometrySmith(N, V, L, alpha_g);
    const float3 f_specular = FHs0 * Ds * Gs / (4.0f * NdotV * NdotL);

    // Clearcoat
    const float Fcc = fresnelSchlickR(LdotH, 0.04f);
    const float alpha_cc = 0.1f + (0.001f - 0.1f) * disney->clearcoat_gloss; // lerp
    const float Dcc = GTR1(NdotH, alpha_cc);
    const float Gcc = geometrySmith(N, V, L, 0.25f);
    const float3 f_clearcoat = make_float3( 0.25f * disney->clearcoat * (Fcc * Dcc * Gcc) / (4.0f * NdotV * NdotL) ); 

    const float3 out = ( 1.0f - disney->metallic ) * ( lerp( f_diffuse, f_subsurface, disney->subsurface ) + f_sheen ) + f_specular + f_clearcoat;
    return out * clamp(NdotL, 0.0f, 1.0f) / math::pi;
}

/**
 * @ref http://simon-kallweit.me/rendercompo2015/report/#adaptivesampling
 * 
 * @todo Investigate correct evaluation of PDF.
 */
extern "C" __device__ float __direct_callable__pdf_disney(SurfaceInteraction* si, void* mat_data)
{
    const DisneyData* disney = reinterpret_cast<DisneyData*>(mat_data);

    const float3 V = -si->wi;
    const float3 L = si->wo;
    const float3 N = si->n;

    const float diffuse_ratio = 0.5f * (1.0f - disney->metallic);
    const float specular_ratio = 1.0f - diffuse_ratio;

    const float NdotL = abs(dot(N, L));
    const float NdotV = abs(dot(N, V));

    const float alpha = fmaxf(0.001f, disney->roughness * disney->roughness);
    const float alpha_cc = 0.1f + (0.001f - 0.1f) * disney->clearcoat_gloss; // lerp
    const float3 H = normalize(V + L);
    const float NdotH = abs(dot(H, N));

    const float pdf_Ds = GTR2(NdotH, alpha);
    const float pdf_Dcc = GTR1(NdotH, alpha_cc);
    const float ratio = 1.0f / (1.0f + disney->clearcoat);
    const float pdf_specular = (pdf_Dcc + ratio * (pdf_Ds - pdf_Dcc)) / (4.0f * NdotH);
    const float pdf_diffuse = NdotL / math::pi;

    return diffuse_ratio * pdf_diffuse + specular_ratio * pdf_specular;
}

// Area emitter ------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__area_emitter(SurfaceInteraction* si, void* surface_data)
{
    const AreaEmitterData* area = reinterpret_cast<AreaEmitterData*>(surface_data);
    si->trace_terminate = true;
    float is_emitted = 1.0f;
    if (!area->twosided)
        is_emitted = dot(si->wi, si->n) < 0.0f ? 1.0f : 0.0f;
    
    si->emission = optixDirectCall<float3, SurfaceInteraction*, void*>(
        area->tex_program_id, si, area->tex_data) * area->strength * is_emitted;
}