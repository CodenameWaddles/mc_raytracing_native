#include "hip/hip_runtime.h"
#include <prayground/optix/cuda/device_util.cuh>
#include <prayground/core/spectrum.h>
#include <prayground/core/ray.h>
#include <prayground/core/onb.h>
#include <prayground/math/random.h>

#include <prayground/material/dielectric.h>
#include <prayground/material/diffuse.h>
#include <prayground/material/disney.h>

#include <prayground/texture/bitmap.h>
#include <prayground/texture/constant.h>
#include <prayground/texture/checker.h>

#include <prayground/shape/trianglemesh.h>
#include <prayground/shape/plane.h>
#include <prayground/shape/sphere.h>

#include <prayground/emitter/envmap.h>

#include "params.h"

using namespace prayground;

// Utilities ------------------------------------------------------------------------------
#define SAMPLE_FUNC(name) __direct_callable__sample_ ## name
#define BSDF_FUNC(name) __continuation_callable__bsdf_ ## name
#define PDF_FUNC(name) __direct_callable__pdf_ ## name

extern "C" { __constant__ LaunchParams params; }

using SurfaceInteraction = SurfaceInteraction_<SampledSpectrum>;

static INLINE DEVICE SurfaceInteraction* getSurfaceInteraction()
{
    const unsigned int u0 = getPayload<0>();
    const unsigned int u1 = getPayload<1>();
    return reinterpret_cast<SurfaceInteraction*>(unpackPointer(u0, u1));
}

__forceinline__ __device__ void traceSpectrum(
    OptixTraversableHandle handle, 
    float3 ro, float3 rd, 
    float tmin, float tmax, 
    unsigned int ray_type, 
    SurfaceInteraction* si, 
    float lambda
)
{
    unsigned int u0, u1;
    packPointer(si, u0, u1);
    optixTrace(
        handle, 
        ro, 
        rd, 
        tmin, 
        tmax, 
        0.0f, 
        OptixVisibilityMask(1), 
        OPTIX_RAY_FLAG_NONE, 
        ray_type, 
        1, 
        ray_type, 
        u0, u1, 
        __float_as_int(lambda)
    );
}

// Raygen ------------------------------------------------------------------------------
static __forceinline__ __device__ void getCameraRay(
    const CameraData& camera, 
    const float& x, const float& y, 
    float3& ro, float3& rd)
{
    rd = normalize(x * camera.U + y * camera.V + camera.W);
    ro = camera.origin;
}

static __forceinline__ __device__ float uniformSpectrumPDF()
{
    return 1.0f / (max_lambda - min_lambda);
}

// Raygen function
extern "C" __global__ void __raygen__spectrum()
{
    const RaygenData* raygen = reinterpret_cast<RaygenData*>(optixGetSbtDataPointer());

    const int subframe_index = params.subframe_index;
    const uint3 idx = optixGetLaunchIndex();
    unsigned int seed = tea<4>(idx.x * params.width + idx.y, subframe_index);

    float radiance;

    int i = params.samples_per_launch;

    // Uniform sampling of lambda
    const float lambda = lerp(min_lambda, max_lambda, rnd(seed));

    do 
    {
        const float2 subpixel_jitter = make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f);
        const float2 d = 2.0f * make_float2(
            (static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(params.width), 
            (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(params.height)
        ) - 1.0f;

        float3 ro, rd;
        getCameraRay(raygen->camera, d.x, d.y, ro, rd);
        
        float throughput = 1.0f;

        SurfaceInteraction si;
        si.seed = seed;
        si.emission = SampledSpectrum{};
        si.albedo = SampledSpectrum{};
        si.trace_terminate = false;
        si.radiance_evaled = false;

        int depth = 0;
        for (;;)
        {
            if (depth >= params.max_depth)
                break;

            traceSpectrum(params.handle, ro, rd, 0.01f, 1e16f, 0, &si, lambda);

            if (si.trace_terminate)
            {
                radiance += si.emission * throughput;
                break;
            }

            // Get emission from area emitter
            if (si.surface_info.type == SurfaceType::AreaEmitter)
            {
                // Evaluating emission from emitter
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.bsdf_id, 
                    &si, 
                    si.surface_info.data
                );
                radiance += si.emission.getSpectrumFromLambda(lambda) * throughput;
                if (si.trace_terminate)
                    break;
            }
            // Specular sampling
            else if (+(si.surface_info.type & SurfaceType::Delta))
            {
                // Samling scattered direction 
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.sample_id, 
                    &si, 
                    si.surface_info.data
                );

                // Evaluate bsdf
                SampledSpectrum bsdf_val = optixContinuationCall<SampledSpectrum, SurfaceInteraction*, void*>(
                    si.surface_info.bsdf_id, 
                    &si, 
                    si.surface_info.data
                );

                throughput *= bsdf_val.getSpectrumFromLambda(lambda);
            }
            // Rough surface sampling with applying MIS
            else if (+(si.surface_info.type & (SurfaceType::Rough | SurfaceType::Diffuse)))
            {
                unsigned int seed = si.seed;
                AreaEmitterInfo light;
                if (params.num_lights > 0)
                {
                    const int light_id = rnd_int(seed, 0, params.num_lights);
                    light = params.lights[light_id];
                }

                const float weight = 1.0f / (params.num_lights + 1);
                float pdf_val = 0.0f;

                // Importance sampling according to the BSDF
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.sample_id, 
                    &si, 
                    si.surface_info.data
                );

                if (rnd(seed) < weight * params.num_lights)
                {
                    // Light sampling
                    float3 to_light = optixDirectCall<float3, AreaEmitterInfo, SurfaceInteraction*>(
                        light.sample_id, 
                        light, 
                        &si
                    );
                    si.wo = normalize(to_light);
                }

                for (int i = 0; i < params.num_lights; i++)
                {
                    // Evaluate PDF of area emitter
                    float light_pdf = optixContinuationCall<float, AreaEmitterInfo, const float3&, const float3&>(
                        params.lights[i].pdf_id,
                        params.lights[i],
                        si.p, 
                        si.wo
                    );
                    pdf_val += weight * light_pdf;
                }

                // Evaluate PDF depends on BSDF 
                float bsdf_pdf = optixDirectCall<float, SurfaceInteraction*, void*>(
                    si.surface_info.pdf_id, 
                    &si, 
                    si.surface_info.data
                );

                pdf_val += weight * bsdf_pdf;

                // Evaluate BSDF
                SampledSpectrum bsdf_val = optixContinuationCall<SampledSpectrum, SurfaceInteraction*, void*>(
                    si.surface_info.bsdf_id, 
                    &si, 
                    si.surface_info.data
                );

                pdf_val = fmaxf(pdf_val, math::eps);

                throughput *= bsdf_val.getSpectrumFromLambda(lambda) / pdf_val;
            }

            ro = si.p;
            rd = si.wo;

            ++depth;
        }
    } while (--i);

    const unsigned int image_idx = idx.x * params.width + idx.y;

    float3 xyz_result = make_float3(
        radiance * CIE_X(lambda) / CIE_Y_integral / uniformSpectrumPDF(),
        radiance * CIE_X(lambda) / CIE_Y_integral / uniformSpectrumPDF(),
        radiance * CIE_X(lambda) / CIE_Y_integral / uniformSpectrumPDF()
    );

    float3 color = XYZToSRGB(xyz_result);

    float3 accum_color = color / static_cast<float>(params.samples_per_launch);

    if (subframe_index > 0)
    {
        const float a = 1.0f / static_cast<float>(subframe_index + 1);
        const float3 accum_color_prev = make_float3(params.accum_buffer[image_idx]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    params.accum_buffer[image_idx] = make_float4(accum_color, 1.0f);
    uchar3 ucolor = make_color(accum_color);
    params.result_buffer[image_idx] = make_uchar4(ucolor.x, ucolor.y, ucolor.z, 255);
}

// Miss function
extern "C" __device__ void __miss__envmap()
{
    MissData* data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    EnvironmentEmitterData* env = reintepret_cast<EnvironmentEmitterData*>(data->env_data);
    SurfaceInteraction* si = getSurfaceInteraction();

    Ray ray = getWorldRay();
    const float lambda = __int_as_float(getPayload<2>());

    const float a = dot(ray.d, ray.d);
    const float half_b = dot(ray.o, ray.d);
    const float c = dot(ray.o, ray.o) - 1e8f*1e8f;
    const float D = half_b * half_b - a * c;

    float sqrtD = sqrtf(D);
    float t = (-half_b + sqrtD) / a;

    const float3 p = normalize(ray.at(t));

    const float phi = atan2(p.z, p.x);
    const float theta = asin(p.y);
    const float u = 1.0f - (phi + math::pi) / (2.0f * math::pi);
    const float v = 1.0f - (theta + math::pi / 2.0f) / math::pi;
    si->uv = make_float2(u, v);
    si->trace_terminate = true;
    si->surface_info.type = SurfaceType::None;
    si->emission = optixDirectCall<SampledSpectrum, SurfaceInteraction*, void*>(
        env->tex_program_id, si, env->tex_data
    );
}

/** 
 * @note Sellmeier equation of BK7 
 * @ref  https://www.thorlabs.co.jp/newgrouppage9.cfm?objectgroup_id=6973&tabname=N-BK7 
 **/
static __forceinline__ __device__ float bk7Index(const float& lambda)
{
    // Convert unit of wavelength: nm -> μm
    const float l = lambda * 0.001f;
    const float l2 = l * l;
    return sqrtf(1.0f + ((1.03961212f * l2) / (l2 - 0.00600069867f)) + ((0.231792344f * l2) / (l2 - 0.0200179144f)) + ((1.01046945 * l2) / (l2 - 103.560653f)));
}

// Material functions
extern "C" __device__ void SAMPLE_FUNC(dielectric)(float lambda, SurfaceInteraction* si, void* mat_data)
{
    const DielectricData* dielectric = reinterpret_cast<DielectricData*>(mat_data);

    float ni = 1.000292f;
    const float lambda = __int_as_float(getPayload<2>());
    float nt = bk7Index(lambda);
    float cosine = dot(si->wi, si->n);
    bool into = cosine < 0;
    float3 outward_normal = into ? si->n : -si->n;

    if (!into) swap(ni, nt);

    cosine = fabs(cosine);
    float sine = sqrtf(1.0f - cosine * cosine);
    bool cannot_refract = ni * sine > nt;

    float reflect_prob = fresnel(cosine, ni, nt);
    unsigned int seed = si->seed;

    if (cannot_refract || reflect_prob > rnd(seed))
        si->wi = reflect(si->wo, outward_normal);
    else
        si->wi = refract(si->wo, outward_normal, cosine, ni, nt);
    si->radiance_evaled = false;
    si->trace_terminate = false;
    si->seed = seed;
}

extern "C" __device__ SampledSpectrum BSDF_FUNC(dielectric)(SurfaceInteraction* si, void* mat_data)
{
    const DielectricData* dielectric = reinterpret_cast<DielectricData*>(mat_data);
    si->emission = make_float3(0.0f);
    return optixDirectCall<SampledSpectrum, SurfaceInteraction*, void*>(dielectric->tex_program_id, si, dielectric->tex_data);
}

extern "C" __device__ float PDF_FUNC(dielectric)(SurfaceInteraction * si, void* mat_data)
{
    return 1.0f;
}

extern "C" __device__ void SAMPLE_FUNC(diffuse)(float lambda, SurfaceInteraction * si, void* mat_data)
{
    const DiffuseData* diffuse = reinterpret_cast<DiffuseData*>(mat_data);

    if (diffuse->twosided)
        si->n = faceforward(si->n, -si->wi, si->n);
    
    si->trace_terminate = false;
    unsigned int seed = si->seed;
    const float z0 = rnd(seed);
    const float z1 = rnd(seed);
    float3 wi = cosineSampleHemisphere(z0, z1);
    Onb onb(si->n);
    onb.inverseTransform(wi);
    si->wi = normalize(wi);
    si->seed = seed;
}

extern "C" __device__ Spectrum BSDF_FUNC(diffuse)(SurfaceInteraction * si, void* mat_data)
{

}

extern "C" __device__ float PDF_FUNC(diffuse)(SurfaceInteraction * si, void* mat_data)
{

}

extern "C" __device__ void SAMPLE_FUNC(disney)(float lambda, SurfaceInteraction * si, void* mat_data)
{

}

extern "C" __device__ Spectrum BSDF_FUNC(disney)(SurfaceInteraction * si, void* mat_data)
{

}

extern "C" __device__ float PDF_FUNC(disney)(SurfaceInteraction * si, void* mat_data)
{

}

// Texture functions
extern "C" __device__ Spectrum DC_FUNC(constant)(SurfaceInteraction * si, void* tex_data)
{
    const BitmapTextureData* image = reinterpret_cast<BitmapTextureData*>(tex_data);
}

extern "C" __device__ Spectrum DC_FUNC(checker)(SurfaceInteraction * si, void* tex_data)
{

}

extern "C" __device__ Spectrum DC_FUNC(bitmap)(SurfaceInteraction * si, void* tex_data)
{

}

// Hitgroup functions
extern "C" __device__ void CH_FUNC(mesh)()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const MeshData* mesh = reinterpret_cast<MeshData*>(data->shape_data);

    Ray ray = getWorldRay();

    const int id = optixGetPrimitiveIndex();
    const Face face = mesh->faces[id];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    const float3 p0 = mesh->vertices[face.vertex_id.x];
    const float3 p1 = mesh->vertices[face.vertex_id.y];
    const float3 p2 = mesh->vertices[face.vertex_id.z];

    const float2 texcoord0 = mesh->texcoords[face.texcoord_id.x];
    const float2 texcoord1 = mesh->texcoords[face.texcoord_id.y];
    const float2 texcoord2 = mesh->texcoords[face.texcoord_id.z];
    const float2 texcoords = (1 - u - v) * texcoord0 + u * texcoord1 + v * texcoord2;

    const float3 n0 = mesh->normals[face.normal_id.x];
    const float3 n1 = mesh->normals[face.normal_id.y];
    const float3 n2 = mesh->normals[face.normal_id.z];

    const float3 local_n = (1 - u - v) * n0 + u * n1 + v * n2;
    const float3 world_n = normalize(optixTransformNormalFromObjectToWorldSpace(local_n));

    auto si = getSurfaceInteraction();
    si->p = ray.at(ray.tmax);
    si->shading.n = world_n;
    si->t = ray.tmax;
    si->wi = ray.d;
    si->uv = texcoords;
    si->surface_info = data->surface_info;

    float3 dpdu, dpdv;
    const float2 duv02 = texcoord0 - texcoord2;
    const float2 duv12 = texcoord1 - texcoord2;
    const float3 dp02 = p0 - p2;
    const float3 dp12 = p1 - p2;
    const float D = duv02.x * duv12.y - duv02.y * duv12.x;
    bool degenerateUV = abs(D) < 1e-8f;
    if (!degenerateUV)
    {
        const float invD = 1.0f / D;
        dpdu = (duv12.y * dp02 - duv02.y * dp12) * invD;
        dpdv = (-duv12.x * dp02 + duv02.x * dp12) * invD;
    }
    if (degenerateUV || length(cross(dpdu, dpdv)) == 0.0f)
    {
        const float3 n = normalize(cross(p2 - p0, p1 - p0));
        Onb onb(n);
        dpdu = onb.tangent;
        dpdv = onb.bitangent;
    }
    si->shading.dpdu = normalize(optixTransformVectorFromObjectToWorldSpace(dpdu));
    si->shading.dpdv = normalize(optixTransformVectorFromObjectToWorldSpace(dpdv));
}

static __forceinline__ __device__ float2 getSphereUV(const float3& p) {
    float phi = atan2(p.z, p.x);
    float theta = asin(p.y);
    float u = 1.0f - (phi + math::pi) / (2.0f * math::pi);
    float v = 1.0f - (theta + math::pi / 2.0f) / math::pi;
    return make_float2(u, v);
}

extern "C" __device__ void IS_FUNC(sphere)()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const SphereData* sphere = reinterpret_cast<SphereData*>(data->shape_data);

    const float3 center = sphere->center;
    const float radius = sphere->radius;
    
    Ray ray = getLocalRay();

    const float3 oc = ray.o - center;
    const float a = dot(ray.d, ray.d);
    const float half_b = dot(oc, ray.d);
    const float c = dot(oc, oc) - radius * radius;
    const float discriminant = half_b * half_b - a * c;

    if (discriminant > 0.0f) {
        float sqrtd = sqrtf(discriminant);
        float t1 = (-half_b - sqrtd) / a;
        bool check_second = true;
        if (t1 > ray.tmin && t1 < ray.tmax) {
            float3 normal = normalize((ray.at(t1) - center) / radius);
            check_second = false;
            optixReportIntersection(t1, 0, float3_as_ints(normal));
        }

        if (check_second) {
            float t2 = (-half_b + sqrtd) / a;
            if (t2 > ray.tmin && t2 < ray.tmax) {
                float3 normal = normalize((ray.at(t2) - center) / radius);
                optixReportIntersection(t2, 0, float3_as_ints(normal));
            }
        }
    }
}

extern "C" __device__ void CH_FUNC(sphere)()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const SphereData* sphere_data = reinterpret_cast<SphereData*>(data->shape_data);

    Ray ray = getWorldRay();

    float3 local_n = getFloat3FromAttribute<0>();
    const float3 world_n = normalize(optixTransformNormalFromObjectToWorldSpace(local_n));

    SurfaceInteraction* si = getSurfaceInteraction();
    si->p = ray.at(ray.tmax);
    si->shading.n = world_n;
    si->t = ray.tmax;
    si->wi = ray.d;
    si->uv = getSphereUV(local_n);
    si->surface_info = data->surface_info;

    float phi = atan2(local_n.z, local_n.x);
    if (phi < 0) phi += 2.0f * math::pi;
    const float theta = acos(local_n.y);
    const float3 dpdu = make_float3(-math::two_pi * local_n.z, 0, math::two_pi * local_n.x);
    const float3 dpdv = math::pi * make_float3(local_n.y * cos(phi), -sin(theta), local_n.y * sin(phi));
    si->shading.dpdu = normalize(optixTransformVectorFromObjectToWorldSpace(dpdu));
    si->shading.dpdv = normalize(optixTransformVectorFromObjectToWorldSpace(dpdv));
}

extern "C" __device__ void IS_FUNC(plane)()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const PlaneData* plane_data = reinterpret_cast<PlaneData*>(data->shape_data);

    const float2 min = plane_data->min;
    const float2 max = plane_data->max;

    Ray ray = getLocalRay();

    const float t = -ray.o.y / ray.d.y;

    const float x = ray.o.x + t * ray.d.x;
    const float z = ray.o.z + t * ray.d.z;

    float2 uv = make_float2((x - min.x) / (max.x - min.x), (z - min.y) / (max.y - min.y));

    if (min.x < x && x < max.x && min.y < z && z < max.y && ray.tmin < t && t < ray.tmax)
        optixReportIntersection(t, 0, float2_as_ints(uv));
}

extern "C" __device__ void CH_FUNC(plane)()
{
    HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());

    Ray ray = getWorldRay();

    float3 local_n = make_float3(0, 1, 0);
    const float3 world_n = normalize(optixTransformNormalFromObjectToWorldSpace(local_n));
    const float2 uv = getFloat2FromAttribute<0>();

    SurfaceInteraction* si = getSurfaceInteraction();

    si->p = ray.at(ray.tmax);
    si->shading.n = world_n;
    si->t = ray.tmax;
    si->wi = ray.d;
    si->uv = uv;
    si->surface_info = data->surface_info;
    si->shading.dpdu = optixTransformNormalFromObjectToWorldSpace(make_float3(1.0f, 0.0f, 0.0f));
    si->shading.dpdv = optixTransformNormalFromObjectToWorldSpace(make_float3(0.0f, 0.0f, 1.0f));
}