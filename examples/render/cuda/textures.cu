#include "hip/hip_runtime.h"
#include "util.cuh"

extern "C" __device__ Vec3f __direct_callable__bitmap(const Vec2f& uv, void* tex_data) {
    const auto* image = reinterpret_cast<BitmapTexture::Data*>(tex_data);
    float4 c = tex2D<float4>(image->texture, uv.x(), uv.y());
    return Vec3f(c);
}

extern "C" __device__ Vec3f __direct_callable__constant(const Vec2f& uv, void* tex_data) {
    const auto* constant = reinterpret_cast<ConstantTexture::Data*>(tex_data);
    return constant->color;
}

extern "C" __device__ Vec3f __direct_callable__checker(const Vec2f& uv, void* tex_data) {
    const auto* checker = reinterpret_cast<CheckerTexture::Data*>(tex_data);
    const bool is_odd = sinf(uv.x() * math::pi * checker->scale) * sinf(uv.y() * math::pi * checker->scale) < 0;
    return lerp(checker->color1, checker->color2, (float)is_odd);
}