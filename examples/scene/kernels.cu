#include "hip/hip_runtime.h"
#include <prayground/prayground.h>
#include "params.h"

extern "C" { __constant__ LaunchParams params; }

using SurfaceInteraction = SurfaceInteraction_<Vec3f>;

static __forceinline__ __device__ SurfaceInteraction* getSurfaceInteraction()
{
	const uint32_t u0 = getPayload<0>();
	const uint32_t u1 = getPayload<1>();
	return reinterpret_cast<SurfaceInteraction*>(unpackPointer(u0, u1));
}

static __forceinline__ __device__ void trace(
	OptixTraversableHandle handle, const Vec3f& ro, const Vec3f& rd,
	float tmin, float tmax, float ray_time, uint32_t ray_type, SurfaceInteraction* si)
{
	uint32_t u0, u1;
	packPointer(si, u0, u1);
	optixTrace(
		handle, ro, rd,
		tmin, tmax, ray_time,
		OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE,
		ray_type, 2, ray_type,
		u0, u1
	);
}

// raygen
extern "C" __device__ void __raygen__pinhole()
{
	const pgRaygenData<Camera>* raygen = (pgRaygenData<Camera>*)optixGetSbtDataPointer();

	const int frame = params.frame;

	const Vec3ui idx(optixGetLaunchIndex());
	uint32_t seed = tea<4>(idx.y() * params.width + idx.x(), frame);

	Vec3f result(0.0f);
	Vec3f normal(0.0f);

	int i = params.samples_per_launch;

	while (i > 0)
	{
		const Vec2f jitter = UniformSampler::get2D(seed) - 0.5f;
		const Vec2f d = 2.0f * Vec2f(
			(static_cast<float>(idx.x()) + jitter.x()) / params.width,
			(static_cast<float>(idx.y()) + jitter.y()) / params.height
		) - 1.0f;

		Vec3f ro, rd;
		getCameraRay(raygen->camera, d.x(), d.y(), ro, rd);

		Vec3f throughput(1.0f);

		SurfaceInteraction si;
		si.seed = seed;
		si.emission = 0.0f;
		si.albedo = 0.0f;
		si.trace_terminate = false;
		si.radiance_evaled = false;

		int depth = 0;
		for (;;)
		{
            if ( depth >= params.max_depth )
				break;

            trace(params.handle, ro, rd, 0.01f, 1e16f, rnd(seed), /* ray_type = */ 0, &si);

            if (si.trace_terminate) {
                result += si.emission * throughput;
                break;
            }

            if (depth == 0)
                normal = si.shading.n;

            // Get emission from area emitter
            if ( si.surface_info.type == SurfaceType::AreaEmitter )
            {
                // Evaluating emission from emitter
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);

                result += si.emission * throughput;
                if (si.trace_terminate)
                    break;
            }
            // Specular sampling
            else if (+(si.surface_info.type & SurfaceType::Delta))
            {
                // Sampling scattered direction
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.sample, &si, si.surface_info.data);
                
                // Evaluate bsdf
                Vec3f bsdf_val = optixContinuationCall<Vec3f, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);
                throughput *= bsdf_val;
            }
            // Rough surface sampling with applying MIS
            else if ( +(si.surface_info.type & (SurfaceType::Rough | SurfaceType::Diffuse)) )
            {
                // Importance sampling according to the BSDF
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.sample, &si, si.surface_info.data);

                // Evaluate PDF of area emitter
                float pdf = optixDirectCall<float, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.pdf, &si, si.surface_info.data);

                // Evaluate BSDF
                Vec3f bsdf = optixContinuationCall<Vec3f, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);

                throughput *= bsdf / pdf;
            }
            
            ro = si.p;
            rd = si.wi;

            ++depth;
		}
	}

    const uint32_t image_idx = idx.y() * params.width + idx.x();

    if (result.x() != result.x()) result.x() = 0.0f;
    if (result.y() != result.y()) result.y() = 0.0f;
    if (result.z() != result.z()) result.z() = 0.0f;

    Vec3f accum_color = result / static_cast<float>(params.samples_per_launch);

    if (frame > 0)
    {
        const float a = 1.0f / static_cast<float>(frame + 1);
        const Vec3f accum_color_prev(params.accum_buffer[image_idx]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    params.accum_buffer[image_idx] = Vec4f(accum_color, 1.0f);
    Vec3u color = make_color(accum_color);
    params.result_buffer[image_idx] = Vec4u(color, 255);
}

// Miss
extern "C" __device__ void __miss__envmap()
{
    pgMissData* data = reinterpret_cast<pgMissData*>(optixGetSbtDataPointer());
    auto* env = reinterpret_cast<EnvironmentEmitter::Data*>(data->env_data);
    SurfaceInteraction* si = getSurfaceInteraction();

    Ray ray = getWorldRay();

    const float a = dot(ray.d, ray.d);
    const float half_b = dot(ray.o, ray.d);
    const float c = dot(ray.o, ray.o) - 1e8f * 1e8f;
    const float discriminant = half_b * half_b - a * c;

    float sqrtd = sqrtf(discriminant);
    float t = (-half_b + sqrtd) / a;

    Vec3f p = normalize(ray.at(t));

    float phi = atan2(p.z(), p.x());
    float theta = asin(p.y());
    float u = 1.0f - (phi + math::pi) / (2.0f * math::pi);
    float v = 1.0f - (theta + math::pi / 2.0f) * math::inv_pi;
    si->shading.uv = Vec2f(u, v);
    si->trace_terminate = true;
    si->surface_info.type = SurfaceType::None;
    si->emission = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        env->texture.prg_id, si, env->texture.data);
}

extern "C" __device__ void __miss__shadow()
{
    setPayload<0>(1);
}

// Hitgroups 
extern "C" __device__ void __intersection__plane()
{
    const pgHitgroupData* data = reinterpret_cast<pgHitgroupData*>(optixGetSbtDataPointer());
    const Plane::Data* plane = reinterpret_cast<Plane::Data*>(data->shape_data);

    const Vec2f min = plane->min;
    const Vec2f max = plane->max;

    Ray ray = getLocalRay();

    const float t = -ray.o.y() / ray.d.y();

    const float x = ray.o.x() + t * ray.d.x();
    const float z = ray.o.z() + t * ray.d.z();

    Vec2f uv(x / (max.x() - min.x()), z / (max.y() - min.y()));

    if (min.x() < x && x < max.x() && min.y() < z && z < max.y() && ray.tmin < t && t < ray.tmax)
        optixReportIntersection(t, 0, Vec3f_as_ints(Vec3f(0, 1, 0)), Vec2f_as_ints(uv));
}

static __forceinline__ __device__ Vec2f getSphereUV(const Vec3f& p) {
    float phi = atan2(p.z(), p.x());
    float theta = asin(p.y());
    float u = 1.0f - (phi + math::pi) / (2.0f * math::pi);
    float v = 1.0f - (theta + math::pi / 2.0f) / math::pi;
    return Vec2f(u, v);
}

extern "C" __device__ void __intersection__sphere() {
    const pgHitgroupData* data = reinterpret_cast<pgHitgroupData*>(optixGetSbtDataPointer());
    const Sphere::Data* sphere = reinterpret_cast<Sphere::Data*>(data->shape_data);

    const Vec3f center = sphere->center;
    const float radius = sphere->radius;

    Ray ray = getLocalRay();

    const Vec3f oc = ray.o - center;
    const float a = dot(ray.d, ray.d);
    const float half_b = dot(oc, ray.d);
    const float c = dot(oc, oc) - radius * radius;
    const float discriminant = half_b * half_b - a * c;

    if (discriminant > 0.0f) {
        float sqrtd = sqrtf(discriminant);
        float t1 = (-half_b - sqrtd) / a;
        bool check_second = true;
        if (t1 > ray.tmin && t1 < ray.tmax) {
            Vec3f normal = normalize((ray.at(t1) - center) / radius);
            Vec2f uv = getSphereUV(normal);
            check_second = false;
            optixReportIntersection(t1, 0, Vec3f_as_ints(normal), Vec2f_as_ints(uv));
        }

        if (check_second) {
            float t2 = (-half_b + sqrtd) / a;
            if (t2 > ray.tmin && t2 < ray.tmax) {
                Vec3f normal = normalize((ray.at(t2) - center) / radius);
                Vec2f uv = getSphereUV(normal);
                optixReportIntersection(t2, 0, Vec3f_as_ints(normal), Vec2f_as_ints(uv));
            }
        }
    }
}

extern "C" __device__ void __closesthit__custom()
{
    pgHitgroupData* data = reinterpret_cast<pgHitgroupData*>(optixGetSbtDataPointer());

    Ray ray = getWorldRay();

    Vec3f local_n = getVec3fFromAttribute<0>();
    Vec3f world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);
    Vec2f uv = getVec2fFromAttribute<3>();

    auto* si = getSurfaceInteraction();

    si->p = ray.at(ray.tmax);
    si->shading.n = world_n;
    si->t = ray.tmax;
    si->wo = ray.d;
    si->shading.uv = uv;
    si->surface_info = data->surface_info;
}

extern "C" __device__ void __closesthit__mesh()
{
    pgHitgroupData* data = reinterpret_cast<pgHitgroupData*>(optixGetSbtDataPointer());
    const TriangleMesh::Data* mesh_data = reinterpret_cast<TriangleMesh::Data*>(data->shape_data);

    Ray ray = getWorldRay();

    const int prim_id = optixGetPrimitiveIndex();
    const Face face = mesh_data->faces[prim_id];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    const Vec3f p0 = mesh_data->vertices[face.vertex_id.x()];
    const Vec3f p1 = mesh_data->vertices[face.vertex_id.y()];
    const Vec3f p2 = mesh_data->vertices[face.vertex_id.z()];

    const Vec2f texcoord0 = mesh_data->texcoords[face.texcoord_id.x()];
    const Vec2f texcoord1 = mesh_data->texcoords[face.texcoord_id.y()];
    const Vec2f texcoord2 = mesh_data->texcoords[face.texcoord_id.z()];
    const Vec2f texcoords = (1 - u - v) * texcoord0 + u * texcoord1 + v * texcoord2;

    const Vec3f n0 = mesh_data->normals[face.normal_id.x()];
    const Vec3f n1 = mesh_data->normals[face.normal_id.y()];
    const Vec3f n2 = mesh_data->normals[face.normal_id.z()];

    // Linear interpolation of normal by barycentric coordinates.
    Vec3f local_n = (1.0f - u - v) * n0 + u * n1 + v * n2;
    Vec3f world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);

    SurfaceInteraction* si = getSurfaceInteraction();
    si->p = ray.at(ray.tmax);
    si->shading.n = world_n;
    si->t = ray.tmax;
    si->wo = ray.d;
    si->shading.uv = texcoords;
    si->surface_info = data->surface_info;
}

extern "C" __device__ void __closesthit__shadow()
{
    setPayload<0>(0);
}

// Surfaces
// Diffuse -----------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_diffuse(SurfaceInteraction * si, void* mat_data) {
    const Diffuse::Data* diffuse = reinterpret_cast<Diffuse::Data*>(mat_data);

    if (diffuse->twosided)
        si->shading.n = faceforward(si->shading.n, -si->wo, si->shading.n);

    si->trace_terminate = false;
    uint32_t seed = si->seed;
    Vec2f u = UniformSampler::get2D(seed);
    Vec3f wi = cosineSampleHemisphere(u[0], u[1]);
    Onb onb(si->shading.n);
    onb.inverseTransform(wi);
    si->wi = normalize(wi);
    si->seed = seed;
}

extern "C" __device__ Vec3f __continuation_callable__bsdf_diffuse(SurfaceInteraction * si, void* mat_data)
{
    const Diffuse::Data* diffuse = reinterpret_cast<Diffuse::Data*>(mat_data);
    const Vec3f albedo = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        diffuse->texture.prg_id, si, diffuse->texture.data);
    si->albedo = albedo;
    si->emission = Vec3f(0.0f);
    const float cosine = fmaxf(0.0f, dot(si->shading.n, si->wi));
    return albedo * cosine * math::inv_pi;
}

extern "C" __device__ float __direct_callable__pdf_diffuse(SurfaceInteraction * si, void* mat_data)
{
    const float cosine = fmaxf(0.0f, dot(si->shading.n, si->wi));
    return cosine * cosine * math::inv_pi;
}

// Dielectric --------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_glass(SurfaceInteraction * si, void* mat_data) {
    const Dielectric::Data* dielectric = reinterpret_cast<Dielectric::Data*>(mat_data);

    float ni = 1.000292f; // air
    float nt = dielectric->ior;  // ior specified 
    float cosine = dot(si->wo, si->shading.n);
    bool into = cosine < 0;
    Vec3f outward_normal = into ? si->shading.n : -si->shading.n;

    if (!into) swap(ni, nt);

    cosine = fabs(cosine);
    float sine = sqrtf(1.0 - cosine * cosine);
    bool cannot_refract = (ni / nt) * sine > 1.0f;

    float reflect_prob = fresnel(cosine, ni, nt);
    unsigned int seed = si->seed;

    if (cannot_refract || reflect_prob > rnd(seed))
        si->wi = reflect(si->wo, outward_normal);
    else
        si->wi = refract(si->wo, outward_normal, cosine, ni, nt);
    si->radiance_evaled = false;
    si->trace_terminate = false;
    si->seed = seed;
}

extern "C" __device__ Vec3f __continuation_callable__bsdf_glass(SurfaceInteraction * si, void* mat_data)
{
    const Dielectric::Data* dielectric = reinterpret_cast<Dielectric::Data*>(mat_data);
    si->emission = Vec3f(0.0f);
    Vec3f albedo = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        dielectric->texture.prg_id, si, dielectric->texture.data);
    si->albedo = albedo;
    return albedo;
}

extern "C" __device__ float __direct_callable__pdf_glass(SurfaceInteraction * si, void* mat_data)
{
    return 1.0f;
}

// Area emitter ------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__area_emitter(SurfaceInteraction * si, void* surface_data)
{
    const AreaEmitter::Data* area = reinterpret_cast<AreaEmitter::Data*>(surface_data);
    si->trace_terminate = true;
    float is_emitted = dot(si->wo, si->shading.n) < 0.0f ? 1.0f : 0.0f;
    if (area->twosided)
    {
        is_emitted = 1.0f;
        si->shading.n = faceforward(si->shading.n, -si->wo, si->shading.n);
    }

    const Vec3f base = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        area->texture.prg_id, si, area->texture.data);
    si->albedo = base;

    si->emission = base * area->intensity * is_emitted;
}

// Textures
extern "C" __device__ Vec3f __direct_callable__bitmap(SurfaceInteraction * si, void* tex_data) {
    const auto* image = reinterpret_cast<BitmapTexture::Data*>(tex_data);
    float4 c = tex2D<float4>(image->texture, si->shading.uv.x(), si->shading.uv.y());
    return Vec3f(c);
}

extern "C" __device__ Vec3f __direct_callable__constant(SurfaceInteraction * si, void* tex_data) {
    const auto* constant = reinterpret_cast<ConstantTexture::Data*>(tex_data);
    return constant->color;
}

extern "C" __device__ Vec3f __direct_callable__checker(SurfaceInteraction * si, void* tex_data) {
    const auto* checker = reinterpret_cast<CheckerTexture::Data*>(tex_data);
    const bool is_odd = sinf(si->shading.uv.x() * math::pi * checker->scale) * sinf(si->shading.uv.y() * math::pi * checker->scale) < 0;
    return lerp(checker->color1, checker->color2, (float)is_odd);
}