#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 

#include <optix.h>
#include <cuda/random.h>
#include "../../optix/util.h"
#include "../../optix/sbt.h"

#include "../../shape/optix/sphere.cuh"
#include "../../shape/optix/trianglemesh.cuh"

#include "../../material/conductor.h"
#include "../../material/dielectric.h"
#include "../../material/diffuse.h"
#include "../../material/emitter.h"

#include "../../texture/checker.h"
#include "../../texture/image.h"

// -------------------------------------------------------------------------------
CALLABLE_FUNC void RG_FUNC(raygen)()
{
	const int w = params.width;
	const int h = params.height;
	const float3 eye = params.eye;
	const float3 U = params.U;
	const float3 V = params.V;
	const float3 W = params.W;
	const uint3 idx = optixGetLaunchIndex();
	const int subframe_index = params.subframe_index;

	unsigned int seed = tea<4>(idx.y * w + idx.x, subframe_index);

	float3 result = make_float3(0.0f, 0.0f, 0.0f);
	int i = params.samples_per_launch;

	do
	{
		const float2 subpixel_jitter = make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f);

		const float2 d = 2.0f * make_float2(
			(static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(w),
			(static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(h)
		) - 1.0f;
		float3 ray_direction = normalize(d.x * U + d.y * V + W);
		float3 ray_origin = eye;

		oprt::SurfaceInteraction si;
		si.seed = seed;
		si.emission = make_float3(0.0f);
		si.radiance = make_float3(0.0f);
		si.attenuation = make_float3(1.0f);
		si.trace_terminate = false;
		si.radiance_evaled = false;

		// float3 radiance = make_float3(1.0f);

		int depth = 0;
		for ( ;; ) {
			trace_radiance(
				params.handle,
				ray_origin, 
				ray_direction, 
				0.01f, 
				1e16f, 
				&si 
			);
			
			if ( !si.radiance_evaled )
				result += si.emission * si.attenuation;
			
			result += si.radiance * si.attenuation;
	
			if ( si.trace_terminate || depth >= params.max_depth ) {
				break;
			}
			
			ray_origin = si.p;
			ray_direction = si.wo;

			++depth;
		}
	} while (--i);

	const uint3 launch_index = optixGetLaunchIndex();
	const unsigned int image_index = launch_index.y * params.width + launch_index.x;
	float3 accum_color = result / static_cast<float>(params.samples_per_launch);

	if (subframe_index > 0)
	{
		const float a = 1.0f / static_cast<float>(subframe_index + 1);
		const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
		accum_color = lerp(accum_color_prev, accum_color, a);
	}
	params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
	params.frame_buffer[image_index] = make_color(accum_color);
}

// -------------------------------------------------------------------------------
CALLABLE_FUNC void MS_FUNC(radiance)()
{
	oprt::MissData* rt_data = reinterpret_cast<oprt::MissData*>(optixGetSbtDataPointer());
	oprt::SurfaceInteraction *si = get_surfaceinteraction();

	// si->radiance = make_float3(rt_data->bg_color);
	si->emission = make_float3(rt_data->bg_color);
	si->trace_terminate = true;
}
