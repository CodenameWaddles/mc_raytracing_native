#include <prayground/core/spectrum.h>
#include <prayground/optix/macros.h>

namespace prayground {

    extern "C" HOST void initRGB2SpectrumTableOnGPU(
        SampledSpectrum& white, 
        SampledSpectrum& cyan, 
        SampledSpectrum& magenta, 
        SampledSpectrum& yellow, 
        SampledSpectrum& red, 
        SampledSpectrum& green,
        SampledSpectrum& blue)
    {
        hipMemcpyToSymbol((void*)rgb2spectrum_white, &white, sizeof(SampledSpectrum));
        hipMemcpyToSymbol((void*)rgb2spectrum_cyan, &cyan, sizeof(SampledSpectrum));
        hipMemcpyToSymbol((void*)rgb2spectrum_magenta, &magenta, sizeof(SampledSpectrum));
        hipMemcpyToSymbol((void*)rgb2spectrum_yellow, &yellow, sizeof(SampledSpectrum));
        hipMemcpyToSymbol((void*)rgb2spectrum_red, &red, sizeof(SampledSpectrum));
        hipMemcpyToSymbol((void*)rgb2spectrum_green, &green, sizeof(SampledSpectrum));
        hipMemcpyToSymbol((void*)rgb2spectrum_blue, &blue, sizeof(SampledSpectrum));
    }

} // namespace prayground