#include "hip/hip_runtime.h"
// Smoothed Particle Hydrodynamics 

#include <prayground/physics/cuda/sph.cuh>
#include <prayground/math/util.h>

namespace prayground {

    DEVICE float cubicSpline(float q)
    {
        if (0.0f <= q <= 0.5f)
            return 6.0f * (pow3(q) - pow2(q)) + 1.0f;
        else if (0.5f < q <= 1.0f)
            return 2.0f * pow3(1.0f - q);
        else
            return 0.0f;
    }

    DEVICE float cubicSplineDerivative(float q)
    {
        if (0.0f <= q <= 0.5f)
            return 6.0f * (3.0f * pow2(q) - 2.0f * q);
        else if (0.5f < q <= 1.0f)
            return -6.0f * pow2(1.0f - q);
        else
            return 0.0f;
    }

    DEVICE float particleKernel(float r, float kernel_size)
    {
        auto q = r / kernel_size;
        auto norm_factor = 8.0f / (math::pi * pow3(kernel_size));
        return norm_factor * cubicSpline(q);
    }

    DEVICE float particleKernelDerivative(float r, float kernel_size)
    {
        auto q = r / kernel_size;
        auto norm_factor = 8.0f / (math::pi * pow3(kernel_size));
        return norm_factor * cubicSplineDerivative(q);
    }

    extern "C" GLOBAL void computeDensity(SPHParticle::Data* particles, uint32_t num_particles, SPHConfig config) 
    {
        // Global thread ID equals particle index i
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_particles) return;

        SPHParticle::Data& pi = particles[idx];

        const float h = config.kernel_size;

        for (auto j = 0; j < num_particles; j++) {
            if (j == idx) continue;

            // Reconstruct density from mass and kernel
            auto pj = particles[j];
            float r = length(pi.position - pj.position);

            // Ignore particles outside of kernel size
            if (r > h) continue;

            pi.density += pj.mass * particleKernel(r, h);
        }
    }

    extern "C" GLOBAL void computePressure(SPHParticle::Data* particles, uint32_t num_particles, SPHConfig config)
    {
        // Global thread ID equals particle index i
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_particles) return;

        SPHParticle::Data& pi = particles[idx];
        pi.pressure = config.stiffness * (pi.density - config.rest_density);
    }

    extern "C" GLOBAL void computeForce(SPHParticle::Data* particles, uint32_t num_particles, SPHConfig config)
    {
        // Global thread ID equals particle index i
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_particles) return;

        SPHParticle::Data& pi = particles[idx];

        Vec3f pressure_force(0.0f);
        Vec3f viscosity_force(0.0f);

        const float h = config.kernel_size;




        for (auto j = 0; j < num_particles; j++) {
            if (j == idx) continue;

            auto pj = particles[j];

            Vec3f pi2pj = pj.position - pi.position;
            float r = length(pi2pj);
            if (r > h) continue;

            viscosity_force += (pj.mass * (pi.velocity - pi.velocity) * 2.0f * particleKernelDerivative(r, h)) / (pj.density * r);

            pressure_force += -pi2pj * pj.mass * (pi.pressure / pow2(pi.density) + (pj.pressure / pow2(pj.density))) * particleKernelDerivative(r, h);
        }
        // Compute viscosity force
        viscosity_force *= -1.0f * pi.mass * pi.velocity;

        // Compute pressure force
        pressure_force *= -1.0f;

        pi.force = pressure_force + viscosity_force + config.external_force;
    }

    extern "C" GLOBAL void updateParticle(SPHParticle::Data* particles, uint32_t num_particles, SPHConfig config)
    {
        // Global thread ID equals particle index i
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_particles) return;

        SPHParticle::Data& pi = particles[idx];

        // Update velocity
        pi.velocity += config.time_step * pi.force / pi.mass;

        // Update position
        pi.position += config.time_step * pi.velocity;
    }

    extern "C" HOST void solveSPH(SPHParticle::Data* d_particles, uint32_t num_particles, SPHConfig config) 
    {
        constexpr int NUM_MAX_THREADS = 1024;
        constexpr int NUM_MAX_BLOCKS = 65536;

        // Determine thread size
        const int num_threads = min((int)num_particles, NUM_MAX_THREADS);
        dim3 threads_per_block(num_threads, 1);

        // Determine block size
        const int num_blocks = num_particles / num_threads + 1;
        dim3 block_dim(num_blocks, 1);

        computeDensity<<<block_dim, threads_per_block>>>(d_particles, num_particles, config);
        computePressure<<<block_dim, threads_per_block>>>(d_particles, num_particles, config);
        computeForce<<<block_dim, threads_per_block>>>(d_particles, num_particles, config);
        updateParticle<<<block_dim, threads_per_block>>>(d_particles, num_particles, config);
    }

} // namespace prayground