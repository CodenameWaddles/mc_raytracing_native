#include "hip/hip_runtime.h"
﻿#include <prayground/prayground.h>
#include "params.h"

extern "C" __constant__ LaunchParams params;

struct SurfaceInteraction {
    Vec3f p;
    Vec3f n;
    Vec3f incoming;
    Vec3f albedo;
    double alpha;
    Vec2f uv;
    SurfaceInfo surface_info;
};

static INLINE DEVICE SurfaceInteraction* getSurfaceInteraction()
{
    const uint32_t u0 = getPayload<0>();
    const uint32_t u1 = getPayload<1>();
    return reinterpret_cast<SurfaceInteraction*>(unpackPointer(u0, u1));
}

INLINE DEVICE void trace(
    OptixTraversableHandle handle, 
    const Vec3f& ro, const Vec3f& rd, 
    float tmin, float tmax, SurfaceInteraction* si)
{
    uint32_t u0, u1;
    packPointer(si, u0, u1);
    optixTrace(
        handle, ro, rd, 
        tmin, tmax, 0.0f, 
        OptixVisibilityMask(2), 
        OPTIX_RAY_FLAG_NONE, 
       (uint32_t)RayType::RADIANCE, (uint32_t)RayType::N_RAY, (uint32_t)RayType::RADIANCE,
        u0, u1);
}

static INLINE DEVICE bool shadowTrace(
  OptixTraversableHandle handle, const Vec3f& ro, const Vec3f& rd,
  float tmin, float tmax, int visibilityMask)
{
  return false;
  uint32_t hit = 0u;
  optixTrace(handle, ro.toCUVec(), rd.toCUVec(),
    tmin, tmax, 0.0f,
    OptixVisibilityMask(visibilityMask), OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
    (uint32_t)RayType::SHADOW, (uint32_t)RayType::N_RAY, (uint32_t)RayType::SHADOW,
    hit);
  return static_cast<bool>(hit);
}

// Raygen -------------------------------------------------------------------------------
extern "C" __device__ void __raygen__pinhole()
{
    const RaygenData* raygen = reinterpret_cast<RaygenData*>(optixGetSbtDataPointer());

    const Vec3ui idx(optixGetLaunchIndex());

    Vec3f color(0.0f);
    Vec3f throughput(1.0f);

    SurfaceInteraction si;
    si.albedo = Vec3f(0.f);
    si.surface_info.type = SurfaceType::None;

    const Vec2f res(params.width, params.height);
    const Vec2f d = 2.0f * (Vec2f(idx.x(), idx.y()) / res) - 1.0f;
    Vec3f ro, rd;
    getCameraRay(raygen->camera, d.x(), d.y(), ro, rd);

    int depth = 0;
    for (;;) {
        break;
      if (depth >= params.max_depth)
        break;

      si.surface_info.type = SurfaceType::None;

      trace(params.handle, ro, rd, 0.00001f, 1e16f, &si); // using miss or closest_hit functions

      si.incoming = rd;

      if (si.surface_info.type == SurfaceType::Diffuse) // Diffuse mesh
      {
        Vec3f radiance = optixContinuationCall<Vec3f, SurfaceInteraction*, void*, const Vec3f&>(
          si.surface_info.callable_id.bsdf, &si, si.surface_info.data, Vec3f{}); // calling __continuation_callable__diffuse_shading

        color += radiance * throughput;
        depth = params.max_depth; // stop !
      }
      else if ((si.surface_info.type == SurfaceType::Reflection) || (si.surface_info.type == SurfaceType::Refraction))
      {
        Vec3f outputDir = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
          si.surface_info.callable_id.sample, &si, si.surface_info.data);
        // calling __direct_callable__reflection_direction or __direct_callable__refraction_direction

        Vec3f Coef = optixContinuationCall<Vec3f, SurfaceInteraction*, void*, const Vec3f&>(
          si.surface_info.callable_id.bsdf, &si, si.surface_info.data, Vec3f{});
        // calling __continuation_callable__reflection_shading or __direct_callable__refraction_direction

        throughput *= Coef;

        ro = si.p;
        rd = outputDir;
        depth++;
      }
    }

    const uint32_t image_index = idx.y() * params.width + idx.x();
    
    Vec3u result = make_color(color);
    params.result_buffer[image_index] = Vec4u(result, 255);
}

// Miss -------------------------------------------------------------------------------
extern "C" __device__ void __miss__envmap()
{
    MissData* data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    const auto* env = reinterpret_cast<EnvironmentEmitter::Data*>(data->env_data);
    auto* si = getSurfaceInteraction();

    Ray ray = getWorldRay();

    const float a = dot(ray.d, ray.d);
    const float half_b = dot(ray.o, ray.d);
    const float c = dot(ray.o, ray.o) - 1e8f*1e8f;
    const float discriminant = half_b * half_b - a*c;

    float sqrtd = sqrtf(discriminant);
    float t = (-half_b + sqrtd) / a;

    Vec3f p = normalize(ray.at(t));

    float phi = atan2(p.z(), p.x());
    float theta = asin(p.y());
    float u = 1.0f - (phi + math::pi) / (2.0f * math::pi);
    float v = 1.0f - (theta + math::pi / 2.0f) * math::inv_pi;
    
    si->uv = Vec2f(u, v);
    si->n = Vec3f(0.0f);
    si->p = p;
    Vec3f color = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        env->texture.prg_id, si, env->texture.data);
    si->albedo = color;
}

// Hitgroups -------------------------------------------------------------------------------

extern "C" __device__ void __closesthit__mesh()
{
    HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const auto* mesh_data = reinterpret_cast<TriangleMesh::Data*>(data->shape_data);

    SurfaceInteraction* si = getSurfaceInteraction();

    Ray ray = getWorldRay();

    const int prim_id = optixGetPrimitiveIndex();
    const Face face = mesh_data->faces[prim_id];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    const Vec2f texcoord0 = mesh_data->texcoords[face.texcoord_id.x()];
    const Vec2f texcoord1 = mesh_data->texcoords[face.texcoord_id.y()];
    const Vec2f texcoord2 = mesh_data->texcoords[face.texcoord_id.z()];
    const Vec2f texcoords = (1 - u - v) * texcoord0 + u * texcoord1 + v * texcoord2;

    Vec3f n0 = normalize(mesh_data->normals[face.normal_id.x()]);
    Vec3f n1 = normalize(mesh_data->normals[face.normal_id.y()]);
    Vec3f n2 = normalize(mesh_data->normals[face.normal_id.z()]);

    // Linear interpolation of normal by barycentric coordinates.
    Vec3f local_n = (1.0f - u - v) * n0 + u * n1 + v * n2;
    Vec3f world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);

    si->p = ray.at(ray.tmax);
    si->n = faceforward(world_n, -ray.d, world_n);
    si->uv = texcoords;
    si->surface_info = data->surface_info;
}

extern "C" __device__ void __closesthit__shadow_mesh() // for shadow rays
{
  // Hit to surface
  setPayload<0>(1);
}

// -------------------------------------------------------------------------------------------------
// Blocks

extern "C" __device__ void __intersection__block()
{
    
}

extern "C" __device__ void __closesthit__block()
{

}

// Diffuse
extern "C" __device__  Vec3f __continuation_callable__diffuse_shading(SurfaceInteraction * si, void* mat_data, const Vec3f & wi)
{
  const auto* diffuse = (Diffuse::Data*)mat_data;
  Vec3f albedo = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
    diffuse->texture.prg_id, si, diffuse->texture.data);
  si->albedo = albedo;

  Vec3f to_light = params.light.pos - si->p;
  const float t_shadow = length(to_light) - 1e-3f;
  const Vec3f light_dir = normalize(to_light);
  // Trace shadow ray
  const bool hit_object = shadowTrace(
    params.handle, si->p, light_dir, 1e-3f, t_shadow, 2);

  Vec3f radiance;
  if (hit_object)
    radiance = 0.2f * si->albedo; //0.2f is the ambient term
  else
    radiance = 0.8f * fmaxf(0.0f, dot(light_dir, si->n)) * si->albedo + 0.2f * si->albedo;

  return radiance;
}

// Reflection
extern "C" __device__  Vec3f __direct_callable__reflection_direction(SurfaceInteraction * si, void* mat_data)
{
  const auto* conductor = (Conductor::Data*)mat_data;
  si->n = faceforward(si->n, -si->incoming, si->n); // Two sided
  return reflect(si->incoming, si->n);
}

extern "C" __device__  Vec3f __continuation_callable__reflection_shading(SurfaceInteraction * si, void* mat_data, const Vec3f & wi)
{
  const auto* conductor = (Conductor::Data*)mat_data;

  Vec3f albedo = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
    conductor->texture.prg_id, si, conductor->texture.data);

  si->albedo = albedo;
  return albedo;
}

// Refraction
extern "C" __device__  Vec3f __direct_callable__refraction_direction(SurfaceInteraction * si, void* mat_data)
{
  const auto* dielectric = (Dielectric::Data*)mat_data;

  float ni = 1.000292f; // air
  float nt = dielectric->ior;  // ior specified 
  float cosine = dot(si->incoming, si->n);
  bool into = cosine < 0;
  Vec3f outward_normal = into ? si->n : -si->n;

  if (!into) swap(ni, nt);

  cosine = fabs(cosine);
  float sine = sqrtf(1.0f - cosine * cosine);
  bool cannot_refract = ni * sine > nt;

  //float reflect_prob = fresnel(cosine, ni, nt);

  Vec3f outgoingDirection;
  if (cannot_refract)
    outgoingDirection = reflect(si->incoming, outward_normal);
  else
    outgoingDirection = refract(si->incoming, outward_normal, cosine, ni, nt);

  return outgoingDirection;
}

extern "C" __device__  Vec3f __continuation_callable__refraction_shading(SurfaceInteraction * si, void* mat_data, const Vec3f & wi)
{
  const auto* dielectric = (Dielectric::Data*)mat_data;
  Vec3f albedo = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
    dielectric->texture.prg_id, si, dielectric->texture.data);
  si->albedo = albedo; 
  // not taking into account Fresnel coefficient 
  //float reflect_prob = fresnel(cosine, ni, nt);
  return albedo;
}


static __forceinline__ __device__ Vec2f getUV(const Vec3f& p) {
    float phi = atan2(p.z(), p.x());
    float theta = asin(p.y());
    float u = 1.0f - (phi + math::pi) / (2.0f * math::pi);
    float v = 1.0f - (theta + math::pi / 2.0f) * math::inv_pi;
    return Vec2f(u, v);
}

extern "C" __device__ Vec3f __direct_callable__bitmap(SurfaceInteraction* si, void* tex_data) {
    const auto* image = reinterpret_cast<BitmapTexture::Data*>(tex_data);
    float4 c = tex2D<float4>(image->texture, si->uv.x(), si->uv.y());
    return Vec3f(c);
}

extern "C" __device__ Vec3f __direct_callable__constant(SurfaceInteraction* si, void* tex_data) {
    const auto* constant = reinterpret_cast<ConstantTexture::Data*>(tex_data);
    return constant->color;
}